#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "error.hpp"

__global__
void interp_u(const double* __restrict__ U_spts, double *U_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const int* __restrict__ out_inds, int nFringe, int nSpts,
    int nVars, int estride, int sstride, int vstride)
{
  const int fpt = (blockDim.x * blockIdx.x + threadIdx.x) / nVars;
  const int var = (blockDim.x * blockIdx.x + threadIdx.x) % nVars;

  if (fpt >= nFringe)
    return;

  int ind = nVars * out_inds[fpt] + var;
  int u_ind = donors[fpt] * estride + var * vstride;
  int w_ind = nSpts * fpt;

  double sum = 0;

  for (int spt = 0; spt < nSpts; spt++)
    sum += weights[w_ind+spt] * U_spts[u_ind + spt*sstride];

  U_out[ind] = sum;
}

void interp_u_wrapper(double *U_spts, double *U_out, int *donors,
    double *weights, int* out_inds, int nFringe, int nSpts, int nVars, int estride,
    int sstride, int vstride, hipStream_t stream_h)
{
  unsigned int threads = 128;
  unsigned int blocks = (nVars * nFringe + threads - 1) / threads;

  interp_u<<<blocks, threads, 0, stream_h>>>(U_spts, U_out, donors, weights, out_inds,
      nFringe, nSpts, nVars, estride, sstride, vstride);

  check_error();
}

template <unsigned int nDims, unsigned int nVars>
__global__
void interp_du(const double* __restrict__ dU_spts, double *dU_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const int* __restrict__ out_inds, int nFringe, int nSpts,
    int estride, int sstride, int vstride, int dstride)
{
  const int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFringe)
    return;

  int u_ind = donors[fpt] * estride;
  int w_ind = nSpts * fpt;

  double sum[nDims][nVars] = {0.0};

  for (int spt = 0; spt < nSpts; spt++)
  {
    double wgt = weights[w_ind + spt];
    for (int dim = 0; dim < nDims; dim++)
      for (int var = 0; var < nVars; var++)
        sum[dim][var] += wgt * dU_spts[u_ind + spt*sstride + dim*dstride + var * vstride];
  }

  for (int dim = 0; dim < nDims; dim++)
  {
    for (int var = 0; var < nVars; var++)
    {
      int ind = nVars * (dim + nDims * out_inds[fpt]) + var;
      dU_out[ind] = sum[dim][var];
    }
  }
}

void interp_du_wrapper(double *dU_spts, double *dU_out, int *donors,
    double *weights, int* out_inds, int nFringe, int nSpts, int nVars,
    int nDims, int estride, int sstride, int vstride, int dstride, hipStream_t stream_h)
{
  unsigned int threads = 128;
  unsigned int blocks = (nFringe + threads - 1) / threads;

  if (nDims == 3)
  {
    if (nVars == 1)
      interp_du<3,1><<<blocks, threads, 0, stream_h>>>(dU_spts, dU_out, donors, weights,
        out_inds, nFringe, nSpts, estride, sstride, vstride, dstride);
    else if (nVars == 5)
      interp_du<3,5><<<blocks, threads, 0, stream_h>>>(dU_spts, dU_out, donors, weights,
        out_inds, nFringe, nSpts, estride, sstride, vstride, dstride);
    else
      FatalError("3D nVars case not recognized (expecting 1 or 5)");
  }
  else
    FatalError("TIOGA support for 3D only currently!");

  check_error();
}
