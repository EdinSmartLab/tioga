#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "error.hpp"

__global__
void interp_u(const double* __restrict__ U_spts, double *U_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const int* __restrict__ out_inds, int nFringe, int nSpts,
    int nVars, int estride, int sstride, int vstride)
{
  const int fpt = (blockDim.x * blockIdx.x + threadIdx.x) / nVars;
  const int var = (blockDim.x * blockIdx.x + threadIdx.x) % nVars;

  if (fpt >= nFringe)
    return;

  int ind = nVars * out_inds[fpt] + var;
  int u_ind = donors[fpt] * estride + var * vstride;
  int w_ind = nSpts * fpt;

  double sum = 0;

  for (int spt = 0; spt < nSpts; spt++)
    sum += weights[w_ind+spt] * U_spts[u_ind + spt*sstride];

  U_out[ind] = sum;

//  if (var == 0 && fpt == 928) printf("FACE 232: cell %d, fpt %d, rho %f\n",donors[fpt],fpt,sum);
//  if (var == 0 && abs(sum-1.)>1e-6) printf("cell %d, fpt %d, rho %f\n",donors[fpt],fpt,sum);
  if ((var == 0) && sum < .99)
  {
    printf("Interp point %d/%d: donor %d, val %f\n",fpt,nFringe,donors[fpt],sum);
    printf("  U_spts[%d,%d] = %f\n",donors[fpt],0,U_spts[u_ind]);
  }
}

void interp_u_wrapper(double *U_spts, double *U_out, int *donors,
    double *weights, int* out_inds, int nFringe, int nSpts, int nVars, int estride,
    int sstride, int vstride, hipStream_t stream_h)
{
  unsigned int threads = 128;
  unsigned int blocks = (nVars * nFringe + threads - 1) / threads;

  interp_u<<<blocks, threads, 0, stream_h>>>(U_spts, U_out, donors, weights, out_inds,
      nFringe, nSpts, nVars, estride, sstride, vstride);

  check_error();
}

template <unsigned int nDims, unsigned int nVars>
__global__
void interp_du(const double* __restrict__ dU_spts, double *dU_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const int* __restrict__ out_inds, int nFringe, int nSpts,
    int estride, int sstride, int vstride, int dstride)
{
  const int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFringe)
    return;

  int u_ind = donors[fpt] * estride;
  int w_ind = nSpts * fpt;

  double sum[nDims][nVars] = {0.0};

  for (int spt = 0; spt < nSpts; spt++)
  {
    double wgt = weights[w_ind + spt];
    for (int dim = 0; dim < nDims; dim++)
      for (int var = 0; var < nVars; var++)
        sum[dim][var] += wgt * dU_spts[u_ind + spt*sstride + dim*dstride + var * vstride];
  }

  for (int dim = 0; dim < nDims; dim++)
  {
    for (int var = 0; var < nVars; var++)
    {
      int ind = nVars * (dim + nDims * out_inds[fpt]) + var;
      dU_out[ind] = sum[dim][var];
    }
  }
}

void interp_du_wrapper(double *dU_spts, double *dU_out, int *donors,
    double *weights, int* out_inds, int nFringe, int nSpts, int nVars,
    int nDims, int estride, int sstride, int vstride, int dstride, hipStream_t stream_h)
{
  unsigned int threads = 128;
  unsigned int blocks = (nFringe + threads - 1) / threads;

  if (nDims == 3 && nVars == 5)
    interp_du<3,5><<<blocks, threads, 0, stream_h>>>(dU_spts, dU_out, donors, weights,
        out_inds, nFringe, nSpts, estride, sstride, vstride, dstride);
  else
    FatalError("TIOGA support for 3D only currently!");

  check_error();
}
