#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

/* --- Handy Vector Operation Macros --- */

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
double DOTCROSS4(const double* __restrict__ c,
                 const double* __restrict__ a1, const double* __restrict__ a2,
                 const double* __restrict__ b1, const double* __restrict__ b2)
{
  double d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  xlist.assign(xlist_h.data(), xlist_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true;
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
double lineSegmentDistance(double *p1, double *p2, double *p3, double *p4, double *dx)
{
  // Get the line equations
  double U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  double V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  double W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  double uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  double vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  double uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  double uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  double vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  double den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  double s = (den < 1e-10) ? 0 : (uv*vw - vv*uw) / den;
  double t = (den < 1e-10) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.), 1.);
  t = fmin(fmax(t, 0.), 1.);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = (p3[i] + t*V[i]) - (p1[i] + s*U[i]);

  double dist = 0;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise */
static
__device__
double triTriDistance(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  if (dist < tol) return 0.;

  /// TODO: optimize (no use of point class)
  // Compute pi2 - N2 * X + d2 = 0
  dPoint V01 = dPoint(T1);
  dPoint V11 = dPoint(T1+3);
  dPoint V21 = dPoint(T1+6);

  dPoint V02 = dPoint(T2);
  dPoint V12 = dPoint(T2+3);
  dPoint V22 = dPoint(T2+6);

  dPoint N2 = (V12-V02).cross(V22-V02); // Plane 2
  N2 /= N2.norm();
  double d2 = -(N2*V02);

  dPoint N1 = (V11-V01).cross(V21-V01); // Plane 1
  N1 /= N1.norm();
  double d1 = -(N1*V01);

  // Signed distances of T1 points to T2's plane
  double d01 = N2*V01 + d2;
  double d11 = N2*V11 + d2;
  double d21 = N2*V21 + d2;

  // Signed distances of T2 points to T1's plane
  double d02 = N1*V02 + d1;
  double d12 = N1*V12 + d1;
  double d22 = N1*V22 + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    /* Approximately coplanar; check if one triangle is inside the other */

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && N2*((V12-V02).cross(V01-V02)) > 0;
    inside = inside && N2*((V02-V22).cross(V01-V22)) > 0;
    inside = inside && N2*((V22-V12).cross(V01-V12)) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && N1*((V11-V01).cross(V02-V01)) > 0;
    inside = inside && N1*((V01-V21).cross(V02-V21)) > 0;
    inside = inside && N1*((V21-V11).cross(V02-V11)) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      dPoint P01 = V01 - N2*d01;
      bool inside = true;
      inside = inside && N2*((V12-V02).cross(P01-V02)) > 0;
      inside = inside && N2*((V02-V22).cross(P01-V22)) > 0;
      inside = inside && N2*((V22-V12).cross(P01-V12)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*(d01);
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      dPoint P11 = V11 - N2*d11;
      bool inside = true;
      inside = inside && N2*((V12-V02).cross(P11-V02)) > 0;
      inside = inside && N2*((V02-V22).cross(P11-V22)) > 0;
      inside = inside && N2*((V22-V12).cross(P11-V12)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*(d11);
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      dPoint P21 = V21 - N2*d21;
      bool inside = true;
      inside = inside && N2*((V12-V02).cross(P21-V02)) > 0;
      inside = inside && N2*((V02-V22).cross(P21-V22)) > 0;
      inside = inside && N2*((V22-V12).cross(P21-V12)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*(d21);
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      dPoint P02 = V02 - N1*d02;
      bool inside = true;
      inside = inside && N1*((V11-V01).cross(P02-V01)) > 0;
      inside = inside && N1*((V01-V21).cross(P02-V21)) > 0;
      inside = inside && N1*((V21-V11).cross(P02-V11)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*(d02);
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      dPoint P12 = V12 - N1*d12;
      bool inside = true;
      inside = inside && N1*((V11-V01).cross(P12-V01)) > 0;
      inside = inside && N1*((V01-V21).cross(P12-V21)) > 0;
      inside = inside && N1*((V21-V11).cross(P12-V11)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*(d12);
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      dPoint P22 = V22 - N1*d22;
      bool inside = true;
      inside = inside && N1*((V11-V01).cross(P22-V01)) > 0;
      inside = inside && N1*((V01-V21).cross(P22-V21)) > 0;
      inside = inside && N1*((V21-V11).cross(P22-V11)) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*(d22);
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  dPoint L = N1.cross(N2);
  L /= L.norm();

  double p0 = L*V01;
  double p1 = L*V11;
  double p2 = L*V21;

  double q0 = L*V02;
  double q1 = L*V12;
  double q2 = L*V22;

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = (L*dt).norm();

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

static
__device__
double triTriDistance2(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const double* V01 = T1;
  const double* V11 = T1+3;
  const double* V21 = T1+6;

  const double* V02 = T2;
  const double* V12 = T2+3;
  const double* V22 = T2+6;

  double N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  double norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  double d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  double d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  double d01 = DOT(N2,V01) + d2;
  double d11 = DOT(N2,V11) + d2;
  double d21 = DOT(N2,V21) + d2;

  double d02 = DOT(N1,V02) + d1;
  double d12 = DOT(N1,V12) + d1;
  double d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      double P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*d01;
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      double P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      double P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      double P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      double P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      double P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  double L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  double p0 = DOT(L,V01);
  double p1 = DOT(L,V11);
  double p2 = DOT(L,V21);

  double q0 = DOT(L,V02);
  double q1 = DOT(L,V12);
  double q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

static
__device__ __forceinline__
dPoint faceNormal(const double* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPoint pt0 = dPoint(&xv[0]);
  dPoint pt1 = dPoint(&xv[3]);
  dPoint pt2 = dPoint(&xv[6]);
  dPoint norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPoint(&xv[9]);
  dPoint norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPoint norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

template<int nSideC, int nSideF>
__device__
double intersectionCheck(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec)
{
  /* --- Prerequisites --- */

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  const int sorderC = nSideC-1;
  const int sorderF = nSideF-1;

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  short TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                       {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  double tol = 1e-9;
  double TC[9], TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,nvert>(exv, bboxC);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<nSideC>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Check outer faces of element for intersection with face
#pragma unroll
  for (int f = 0; f < 6; f++)
  {
#pragma unroll
    for (int g = 0; g < sorderC*sorderC; g++)
    {
      int I, J, K;
      switch (f)
      {
        case 0: // Bottom
          I = g / sorderC;
          J = g % sorderC;
          K = 0;
          break;
        case 1: // Top
          I = g / sorderC;
          J = g % sorderC;
          K = sorderC - 1;
          break;
        case 2: // Left
          I = 0;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 3: // Right
          I = sorderC - 1;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 4: // Front
          I = g / sorderC;
          J = 0;
          K = g % sorderC;
          break;
        case 5: // Back
          I = g / sorderC;
          J = sorderC - 1;
          K = g % sorderC;
          break;
      }

      int i0 = I+nSideC*(J+nSideC*K);
      int j0 = i0 + nSideC*nSideC;
      int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
      for (int i = 0; i < 8; i++)
        lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

      // Get triangles for the sub-hex of the larger curved hex
      for (int i = f; i < f+2; i++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = lin2curv[TriPts[i][p]];
          for (int d = 0; d < 3; d++)
            TC[3*p+d] = exv[3*ipt+d];
        }

        cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
        double btol = .05*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
        btol = fmin(btol, minDist);
        if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;

        for (int M = 0; M < sorderF; M++)
        {
          for (int N = 0; N < sorderF; N++)
          {
            int m0 = M + nSideF*N;
            int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
            for (int m = 0; m < 2; m++)
              for (int n = 0; n < 3; n++)
                TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

            // Intersection check between element face tris & cutting-face tris
            for (int j = 0; j < 2; j++)
            {
              for (int p = 0; p < 3; p++)
              {
                int ipt = TriPtsF[j][p];
                for (int d = 0; d < 3; d++)
                  TF[3*p+d] = fxv[3*ipt+d];
              }

              double vec[3];
              double dist = triTriDistance2(TF, TC, vec, tol);

              if (dist < tol)
                return 0.;

              if (dist < minDist)
              {
                for (int d = 0; d < 3; d++)
                  minVec[d] = vec[d];
                minDist = dist;
              }
            }
          }
        }
      }
    }
  }

  if (minDist == BIG_DOUBLE) // Definitely no intersection; use centroids to get vector
  {
    double tmp[3];
    cuda_funcs::getCentroid<3,nvert>(exv,minVec);
    cuda_funcs::getCentroid<3,nvertf>(fxv,tmp);

    minDist = 0;
    for (int d = 0; d < 3; d++)
    {
      minVec[d] -= tmp[d];
      minDist += minVec[d]*minVec[d];
    }

    return sqrt(minDist);
  }

  return minDist;
}

template<int nDims, int nSideC, int nSideF>
__global__
void fillCutMap(dMeshBlock mb, dvec<double> cutFaces, int nCut,
                int* __restrict__ cutFlag, int cutType)
{
  int ic = blockIdx.x * blockDim.x + threadIdx.x;

  if (ic >= mb.ncells) return;
//  if (ic >= 1) return;

  // Figure out how many threads are left in this block after ic>ncells returns
  int blockSize = min(blockDim.x, mb.ncells - blockIdx.x * blockDim.x);

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  int myFlag = DC_UNASSIGNED;
  double myDist = BIG_DOUBLE;
  double myNorm[3] = {0., 0., 0.};
  double myDot;
  double nMin = 0;

  double xv[nDims*nvert];
  __shared__ double fxv[nDims*nvertf];

  // Load up the cell nodes into an array
  for (int i = 0; i < nvert; i++)
  {
    for (int d = 0; d < nDims; d++)
      xv[nDims*i+d] = mb.coord[ic+mb.ncells*(d+nDims*i)]; /// NOTE: 'row-major' ZEFR layout
  }

  double bboxC[2*nDims], bboxF[2*nDims];

  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);

  // btol == 10 times the average side length of the cell's bounding box
  const double btol = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]); // / nDims * 10.
  const double dtol = 1e-3*btol;

  int stride = nDims*nvertf;

  for (int ff = 0; ff < nCut; ff++)
  {
    __syncthreads();

    for (int i = threadIdx.x; i < stride; i += blockSize)
    {
      fxv[i] = cutFaces[ff*stride+i];
    }

    __syncthreads();

    if (myFlag == DC_CUT) continue;

    /*if (mb.rrot)
      getBoundingBox(&cutFaces[ff*stride], nvertf, nDims, bbox, Rmat.data());
    else*/
    cuda_funcs::getBoundingBox<nDims,nvertf>(fxv, bboxF);

    if (myFlag != DC_CUT && cuda_funcs::boundingBoxCheck<nDims>(bboxC, bboxF, btol))
    {
      // Find distance from face to cell
      dPoint vec;
      double dist = intersectionCheck<nSideC,nSideF>(mb, fxv, xv, &vec[0]);
      vec /= dist;

      dPoint norm = faceNormal(fxv);

      if (dist < 1e-8*btol) // They intersect
      {
        myFlag = DC_CUT;
        myDist = 0.;
      }
      else if (myFlag == DC_UNASSIGNED || dist < (myDist - dtol))
      {
        // Unflagged cell, or have a closer face to use
        if (cutType == 0) norm *= -1;

        double dot = norm*vec;

        myDist = dist;
        for (int d = 0; d < 3; d++)
          myNorm[d] = norm[d];
        myDot = dot;
        nMin = 1;

        if (dot < 0) /// TODO: decide on standard orientation
          myFlag = DC_HOLE; // outwards normal = inside cutting surface
        else
          myFlag = DC_NORMAL;
      }
      else if (fabs(dist - myDist) <= dtol)
      {
        // Approx. same dist. to two faces; avg. their normals to decide
        if (cutType == 0) norm *= -1;

          myDist = dist;
          for (int d = 0; d < 3; d++)
            myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
          nMin++;

          myDot = norm*vec;

          if (myDot < 0)
            myFlag = DC_HOLE; // outwards normal = inside cutting surface
          else
            myFlag = DC_NORMAL;
      }
      // else dist > myDist, ignore
    }
  }

//  if (myFlag == DC_CUT)
//    myFlag = (cutType == 1) ? DC_HOLE : DC_NORMAL;

  cutFlag[ic] = myFlag;
}

//void dMeshBlock::directCut(dvec<double> &cutFaces, int nCut, int nvertf, dCutMap &cutMap, int cutType)
void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, int* cutFlag, int cutType)
{
  // Setup cutMap TODO: create initialization elsewhere?
  dvec<int> cutFlag_d;
  cutFlag_d.resize(ncells);

  dvec<double> cutFaces;
  cutFaces.assign(cutFaces_h, nCut*nvertf*nDims);

  int threads = 32;
  int blocks = (ncells + threads - 1) / threads;

  int nbShare = sizeof(double)*nvertf*nDims;

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);
  }

  switch(nvertf)
  {
    case 4:
      switch(nvert)
      {
        case 8:
          fillCutMap<3,2,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
          break;
//        case 27:
//          fillCutMap<3,3,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
        case 64:
          fillCutMap<3,4,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
          break;
//        case 125:
//          fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
        default:
          printf("nvert = %d\n",nvert);
          ThrowException("nvert case not implemented for directCut on device");
      }
      break;
//    case 9:
//      switch(nvert)
//      {
//        case 8:
//          fillCutMap<3,2,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 27:
//          fillCutMap<3,3,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 64:
//          fillCutMap<3,4,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 125:
//          fillCutMap<3,5,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        default:
//          printf("nvert = %d\n",nvert);
//          ThrowException("nvert case not implemented for directCut on device");
//      }
//      break;
    case 16:
      switch(nvert)
      {
        case 8:
          fillCutMap<3,2,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
          break;
//        case 27:
//          fillCutMap<3,3,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
        case 64:
          fillCutMap<3,4,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
          break;
//        case 125:
//          fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
        default:
          printf("nvert = %d\n",nvert);
          ThrowException("nvert case not implemented for directCut on device");
      }
      break;
//    case 25:
//      switch(nvert)
//      {
//        case 8:
//          fillCutMap<3,2,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 27:
//          fillCutMap<3,3,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 64:
//          fillCutMap<3,4,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        case 125:
//          fillCutMap<3,5,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType);
//          break;
//        default:
//          printf("nvert = %d\n",nvert);
//          ThrowException("nvert case not implemented for directCut on device");
//      }
//      break;
  }

  check_error();

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  cutFaces.free_data();
  cutFlag_d.free_data();
}
