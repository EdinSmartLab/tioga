#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  xlist.assign(xlist_h.data(), xlist_h.size());
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true;
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}
