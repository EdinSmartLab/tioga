#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

#include "hip/device_functions.h"
#include "math.h"

#define MAX_UCHAR 255

/* --- Handy Vector Operation Macros --- */

#define NF1 32 // 20-32 depending on unstructured-ness of grid & desire for robustness
#define NF2  4 // 3-6 depending on unstructured-ness of grid & desire for robustness

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
double DOTCROSS4(const double* __restrict__ c,
                 const double* __restrict__ a1, const double* __restrict__ a2,
                 const double* __restrict__ b1, const double* __restrict__ b2)
{
  double d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}

/* --- Misc. Helpful CUDA kernels --- */

__device__ __forceinline__
void print_nodes(const double* pts, int id, int npts)
{
  int idx = threadIdx.x;
  for (int tid = 0; tid < 32; tid++)
  {
    if (idx == tid)
    {
      printf("Points%d = [",id);
      for (int i = 0; i < npts - 1; i++)
        printf("%f %f %f;\n",pts[3*i+0],pts[3*i+1],pts[3*i+2]);

      int I = npts-1;
      printf("%f %f %f];\n",pts[3*I+0],pts[3*I+1],pts[3*I+2]);
    }
  }
}

#define WARP_SZ 32

__device__
inline int lane_id(void) { return threadIdx.x % WARP_SZ; }

__device__
inline int warp_bcast(int v, int leader) { return __shfl(v, leader); }

static inline __device__
float warpAllReduceMin(float val) {
  for (int mask = warpSize/2; mask > 0; mask /= 2)
    val = fminf(val, __shfl_xor(val, mask));
  return val;
}

__device__
int floatToOrderedInt(float floatVal)
{
  int intVal = __float_as_int(floatVal);

  return (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF;
}

__device__
unsigned int floatToUint(float fval)
{
  unsigned int ival = __float_as_uint(fval);
  unsigned int mask = -int(ival >> 31) | 0x80000000;
  return ival ^ mask;
}

__device__
float uintToFloat(unsigned int ival)
{
  unsigned int mask = ((ival >> 31) - 1) | 0x80000000;
  return __uint_as_float(ival ^ mask);
}

__device__
float orderedIntToFloat(int intVal)
{
  return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF );
}

__device__ float atomicMaxf(float* address, float val)
{
  //int *iaddr = (int*)address;
  int old = __float_as_int(*address);
  int assumed;
  while (val > __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

__device__ float atomicMinf(float* address, float val)
{
  int old = __float_as_int(*address);
  int assumed;
  while (val < __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

/*! Warp-aggregated atomic increment
 *  https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/ */
__device__
int atomicAggInc(int *ctr)
{
  int mask = __ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  int res;
  if (lane_id() == leader)
    res = atomicAdd(ctr, __popc(mask));
  // brodcast result
  res = warp_bcast(res, leader);
  // each thread computes its own value
  return res + __popc(mask & ((1 << lane_id()) - 1));
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  xlist.assign(xlist_h.data(), xlist_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::assignHoleMap(bool hasWall, int* nx, int* sam, double* extents)
{
  if (hasWall)
  {
    int size = nx[0]*nx[1]*nx[2];

    std::vector<char> tmp_sam(size);
    for (int i = 0; i < size; i++)
      tmp_sam[i] = (char)sam[i];

    double dx[3];
    for (int d = 0; d < 3; d++)
      dx[d] = (extents[d+3] - extents[d]) / nx[d];

    hm_sam.assign(tmp_sam.data(), size);
    hm_extents.assign(extents, 6);
    hm_nx.assign(nx, 3);
    hm_dx.assign(dx, 3);
  }
  else
  {
    clearHoleMap();
  }
}

void dMeshBlock::clearHoleMap(void)
{
  int nx[3] = {0,0,0};
  double dx[3] = {0,0,0};
  double extents[6] = {0,0,0,0,0,0};

  hm_sam.resize(0);

  hm_nx.assign(nx, 3);
  hm_dx.assign(dx, 3);
  hm_extents.assign(extents, 6);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true;
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
double lineSegmentDistance(double *p1, double *p2, double *p3, double *p4, double *dx)
{
  // Get the line equations
  double U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  double V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  double W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  double uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  double vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  double uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  double uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  double vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  double den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  double s = (den < 1e-10) ? 0 : (uv*vw - vv*uw) / den;
  double t = (den < 1e-10) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.), 1.);
  t = fmin(fmax(t, 0.), 1.);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  double dist = 0;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise 
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
double triTriDistance2(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const double* V01 = T1;
  const double* V11 = T1+3;
  const double* V21 = T1+6;

  const double* V02 = T2;
  const double* V12 = T2+3;
  const double* V22 = T2+6;

  double N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  double norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  double d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  double d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  double d01 = DOT(N2,V01) + d2;
  double d11 = DOT(N2,V11) + d2;
  double d21 = DOT(N2,V21) + d2;

  double d02 = DOT(N1,V02) + d1;
  double d12 = DOT(N1,V12) + d1;
  double d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      double P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      double P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      double P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      double P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      double P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      double P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  double L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  double p0 = DOT(L,V01);
  double p1 = DOT(L,V11);
  double p2 = DOT(L,V21);

  double q0 = DOT(L,V02);
  double q1 = DOT(L,V12);
  double q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

static
__device__ __forceinline__
dPoint faceNormal(const double* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPoint pt0 = dPoint(&xv[0]);
  dPoint pt1 = dPoint(&xv[3]);
  dPoint pt2 = dPoint(&xv[6]);
  dPoint norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPoint(&xv[9]);
  dPoint norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPoint norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}


template<int nSideC, int nSideF>
__device__
double intersectionCheckOne(dMeshBlock &mb, const double* __restrict__ fxv,
    double* __restrict__ minVec, double* TC)
{
  /* --- Prerequisites --- */

  const int sorderF = nSideF-1;

  double tol = 1e-9;
  double TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  for (int M = 0; M < sorderF; M++)
  {
    for (int N = 0; N < sorderF; N++)
    {
      int m0 = M + nSideF*N;
      int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
      for (int m = 0; m < 2; m++)
        for (int n = 0; n < 3; n++)
          TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

template<int nSideC, int nSideF>
__device__
double intersectionCheck(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec)
{
  /* --- Prerequisites --- */

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  const int sorderC = nSideC-1;
  const int sorderF = nSideF-1;

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                       {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  double tol = 1e-9;
  double TC[9], TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,nvert>(exv, bboxC);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<nSideC>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Check outer faces of element for intersection with face
#pragma unroll
  for (int f = 0; f < 6; f++)
  {
#pragma unroll
    for (int g = 0; g < sorderC*sorderC; g++)
    {
      int I, J, K;
      switch (f)
      {
        case 0: // Bottom
          I = g / sorderC;
          J = g % sorderC;
          K = 0;
          break;
        case 1: // Top
          I = g / sorderC;
          J = g % sorderC;
          K = sorderC - 1;
          break;
        case 2: // Left
          I = 0;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 3: // Right
          I = sorderC - 1;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 4: // Front
          I = g / sorderC;
          J = 0;
          K = g % sorderC;
          break;
        case 5: // Back
          I = g / sorderC;
          J = sorderC - 1;
          K = g % sorderC;
          break;
      }

      int i0 = I+nSideC*(J+nSideC*K);
      int j0 = i0 + nSideC*nSideC;
      int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
      for (int i = 0; i < 8; i++)
        lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

      // Get triangles for the sub-hex of the larger curved hex
      for (int i = f; i < f+2; i++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = lin2curv[TriPts[i][p]];
          for (int d = 0; d < 3; d++)
            TC[3*p+d] = exv[3*ipt+d];
        }

        cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
        double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
        btol = fmin(btol, minDist);
        if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;

        for (int M = 0; M < sorderF; M++)
        {
          for (int N = 0; N < sorderF; N++)
          {
            int m0 = M + nSideF*N;
            int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
            for (int m = 0; m < 2; m++)
              for (int n = 0; n < 3; n++)
                TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

            // Intersection check between element face tris & cutting-face tris
            for (int j = 0; j < 2; j++)
            {
              for (int p = 0; p < 3; p++)
              {
                int ipt = TriPtsF[j][p];
                for (int d = 0; d < 3; d++)
                  TF[3*p+d] = fxv[3*ipt+d];
              }

              double vec[3];
              double dist = triTriDistance2(TF, TC, vec, tol);

              if (dist < tol)
                return 0.;

              if (dist < minDist)
              {
                for (int d = 0; d < 3; d++)
                  minVec[d] = vec[d];
                minDist = dist;
              }
            }
          }
        }
      }
    }
  }

  // 3) Definitely no intersection; use centroids to get vector
  if (minDist == BIG_DOUBLE)
  {
    double tmp[3];
    cuda_funcs::getCentroid<3,nvert>(exv,minVec);
    cuda_funcs::getCentroid<3,nvertf>(fxv,tmp);

    minDist = 0;
    for (int d = 0; d < 3; d++) // Vector is face -> cell
    {
      minVec[d] -= tmp[d];
      minDist += minVec[d]*minVec[d];
    }

    return sqrt(minDist);
  }

  return minDist;
}

__device__ __forceinline__
double intersectionCheckLinear(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, const double* __restrict__ bboxC,
    double* __restrict__ minVec, char &cornerOut)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  double tol = 1e-8f;
  double TC[9], TF[9];
  double minDist = 1e15;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxF[6];
  cuda_funcs::getBoundingBox<3,4>(fxv, bboxF);

  double xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
//  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
//  {
//    double rst[3];
//    if (mb.getRefLoc<2>(exv, bboxC, fxv, rst)) ///
//      return 0.;
//  }

  // 2) Find nearest corner of element to face; check only that half of element

  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    double dist = 0.;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        minVec[d] = exv[3*i+d] - xcf[d];
      corner = i;
    }
  }

  cornerOut = corner;

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const int fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

  // 3) Check those faces of element for intersection with face
#pragma unroll
  for (int F = 0; F < 3; F++)
  {
    int f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        int ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      const int TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

template<int nDims, int nSideC, int nSideF>
__global__
void fillCutMap(dMeshBlock mb, dvec<double> cutFaces, int nCut, int nFiltF,
    dvec<int> filt_faces, int* __restrict__ cutFlag, int cutType,
    int* __restrict__ list, int ncells, dvec<int> faceListOut)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= ncells) return;

  int ic = list[tid];  // Get filtered cell ID

  // Figure out how many threads are left in this block after ic>=ncells returns
  int blockSize = min(blockDim.x, ncells - blockIdx.x * blockDim.x);

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;

  const bool LINEAR = (nSideC == 2 && nSideF == 2);

  //bool PRINT = threadIdx.x == 0 && blockIdx.x == 0;

  // Load up the cell nodes into an array
  double xv[nDims*nvert];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < nDims; d++)
      xv[nDims*i+d] = mb.coord[ic+mb.ncells*(d+nDims*i)]; /// NOTE: 'row-major' ZEFR layout

  double bboxC[2*nDims];
  double xcC[3];
  double xcFf[3];
  float oobb[15];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv,xcC);

  bool PRINT = tid == 0 && ic == 865; //(fabs(xcC[0]-.71) < .08) && (fabs(xcC[1]+.56) < .08) && (fabs(xcC[2]+.56) < .08);

  cuda_funcs::getOOBB<nDims,nvert>(xv,oobb,PRINT);

  /*float xcRc[3];
  for (int d = 0; d < 3; d++)
    xcRc[d] = 0.5f * (oobb[9+d] + oobb[12+d]);*/

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 2e-1*href;

  /* --- Pass 1: Coarse-Grained Distance Calculation Using Bounding Boxes --- */

  double distList[NF1];
  int faceList[NF1];
  short nHit = 0;
  __shared__ double fxv_s[nDims*4];
  float fxv_r[nDims*4] = {0.0f};

  for (int i = 0; i < NF1; i++)
  {
    distList[i] = BIG_DOUBLE;
    faceList[i] = -1;
  }

  for (int i = 0; i < nFiltF; i++)
  {
    int ff = filt_faces[i];

    __syncthreads();

    // Stick to just linear component of face [4 corners]
    for (int i = threadIdx.x; i < 4*nDims; i += blockSize)
      fxv_s[i] = cutFaces[ff*stride+i];

    __syncthreads();

    // Transform face points to element OOBB axes
    for (int i = 0; i < 12; i++)
      fxv_r[i] = 0.f;

    for (int k = 0; k < 4; k++)
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
          fxv_r[3*k+i] += oobb[3*i+j] * fxv_s[3*k+j];

    // Get OBB of face in element's axes
    float obbF[6];
    cuda_funcs::getBoundingBox<nDims,4>(fxv_r,obbF);

    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcFf);

    /*
    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcF);

    double dist2 = 0.;
    for (int d = 0; d < 3; d++)
      dist2 += (xcC[d] - xcF[d]) * (xcC[d] - xcF[d]);*/

    float dist1 = cuda_funcs::boundingBoxDist<3>(&oobb[9], &obbF[0]);
    //bool check = cuda_funcs::boundingBoxCheck<3>(oobb,obbF,.01f*(float)dtol);
    bool check = (dist1 <= .01f*(float)dtol);

    double dist2 = 0;
    for (int d = 0; d < 3; d++)
      dist2 += (xcFf[d] - xcC[d]) * (xcFf[d] - xcC[d]);
    dist2 = 0.5*(sqrt(dist2) + dist1);

    if (check)
    {
      // Build up list of faces with bbox intersect from the front
      nHit++;

      int ind = min(NF1 - 1, nHit);

      if (dist2 > distList[ind]) continue;

      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > 0)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }
    else
    {
      int ind = NF1 - 1;

      if (dist2 > distList[ind]) continue;

      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > nHit)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
//      if (PRINT)
//      {
//        printf("Cell 865: faceList = %d %d %d %d %d %d %d %d %d %d %d %d\n",faceList[0],faceList[1],faceList[2],faceList[3],
//            faceList[4],faceList[5],faceList[6],faceList[7],
//            faceList[8],faceList[9],faceList[10],faceList[11]);
//        printf("Cell 865: distList = %.4e %.4e %.4e %.4e %.4e %.4e %.4e %.4e %.4e %.4e %.4e %.4e\n",distList[0],distList[1],distList[2],distList[3],
//            distList[4],distList[5],distList[6],distList[7],
//            distList[8],distList[9],distList[10],distList[11]);
//      }
    }
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

  int ncheck = (nHit > 0) ? min(nHit,NF1) : NF2;

  int faces[NF2];
  for (int i = 0; i < NF2; i++)
  {
    faces[i] = -1;
    distList[i] = 1e15;
  }

  double fxv[nDims*nvertf];
  double myNorm[3] = {0., 0., 0.};
  double myDist = BIG_DOUBLE;
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;

  for (int F = 0; F < ncheck; F++)
  {
    if (LINEAR && myFlag == DC_CUT) continue;

    int ff = faceList[F];
    if (ff < 0)
      continue;

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    dPoint vec;
    double dist = intersectionCheck<2,2>(mb, fxv, xv, &vec[0]);
    //double dist = intersectionCheckLinear(mb, fxv, xv, &vec[0]);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (LINEAR && dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .1*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .1*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from surface; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }

    if (!LINEAR)
    {
      // Insert face into final sorted list of faces to check more fully
      if (dist < distList[NF2-1])
      {
        // Insert and sort
        distList[NF2-1] = dist;
        faces[NF2-1] = ff;
        int ind = NF2 - 1;
        while (distList[ind] < distList[ind-1] && ind > 0)
        {
          swap(distList[ind], distList[ind-1]);
          swap(faces[ind], faces[ind-1]);
          ind--;
        }
      }
    }
  }


  if (!LINEAR)
  {
    for (int i = 0; i < NF2; i++)
      faceListOut[tid*NF2+i] = faces[i];
  }


  /* --- 6/24/17
     + Do 'intersectionCheck()' on linear component of cell/face only
     ++ Do for all ncheck faces
     + If nSideC and nSideF == 2, return this result
     + If nSideC or nSideF > 2, do 'full' intersectionCheck() using
       the nearest half of the element
     --- */

  // If entirely linear grid system, call it good & return
  if (LINEAR)
  {
    if (myFlag == DC_CUT)
      myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

    cutFlag[ic] = myFlag;
    return;
  }

//  if (tid == 0)
//  {
//    printf("%%Cell %d - cell vertices:\n",ic);
//    printf("EXV = [");
//    for (int i = 0; i < nvert-1; i++)
//      printf("%f  %f  %f;\n",xv[3*i],xv[3*i+1],xv[3*i+2]);
//    int I = nvert-1;
//    printf("%f  %f  %f];\n",xv[3*I],xv[3*I+1],xv[3*I+2]);
//    printf("%%Cell %d - Face vertices:\n",ic);
//    for (int F = 0; F < NF2; F++)
//    {
//      int ff = faceListOut[tid*NF2+F];

//      // Load face vertices
//      for (int j = 0; j < stride; j++)
//        fxv[j] = cutFaces[ff*stride+j];

//      printf("%%Face %d:\nFace%d = [",ff,F);
//      for (int j = 0; j < nvertf-1; j++)
//        printf("%f  %f  %f;\n",fxv[3*j],fxv[3*j+1],fxv[3*j+2]);

//      int I = nvertf-1;
//      printf("%f  %f  %f];\n",fxv[3*I],fxv[3*I+1],fxv[3*I+2]);
//    }
//  }
}

template<int nDims, int nSideC, int nSideF>
__global__
void cuttingPass1(dMeshBlock mb, dvec<double> cutFaces, int nFiltF,
    dvec<int> filt_faces, int* __restrict__ cutFlag, int cutType,
    int* __restrict__ list, int ncells, dvec<int> faceListOut, dvec<char> cornerOut)
{
  const int tid = blockIdx.x;
  const int idx = threadIdx.x;
  /// USING 32 THREADS PER BLOCK - 1 CELL PER BLOCK

  if (tid >= ncells) return;

  const int ic = list[tid];  // Get filtered cell ID

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;

  const bool LINEAR = (nSideC == 2 && nSideF == 2);

  //bool PRINT = threadIdx.x == 0 && blockIdx.x == 0;

  __shared__ double xv[nDims*nvert];
//  __shared__ int nHit[1];
  __shared__ float dists[32];
  __shared__ int faces[32];
  __shared__ char corners[32];

  __syncthreads();

  // Load up the cell nodes into an array
  for (int i = idx; i < nvert*nDims; i += blockDim.x)
  {
    int d = i % 3;
    int v = i / 3;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  __syncthreads();

  int myFaces[32];  
  float myDists[32];
  char myCorners[32];

  for (int i = 0; i < 32; i++)
  {
    myFaces[i] = -1;
    myDists[i] = BIG_FLOAT;
    myCorners[i] = -1;
  }

  double bboxC[2*nDims];
  double xcC[3];
  double xcFf[3];
  float oobb[15];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv, xcC);

  bool PRINT = sqrt(xcC[0]*xcC[0]+xcC[1]*xcC[1]+xcC[2]*xcC[2]) > .76;//(abs(xcC[0]+.315) < .03 && abs(xcC[1]+.065)<.03 && abs(xcC[2]+.76)<.03); //tid == 0 && ic == 865; /// DEBUGGING
//if (PRINT) printf("Check cell ID %d\n",ic);
//if (PRINT && idx == 0) print_nodes(xv,idx,8);
//if (PRINT) print_nodes(fxv,ff,4);
  cuda_funcs::getOOBB<nDims,nvert>(xv,oobb,PRINT);

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 2e-1*href;

  /* --- Pass 1: Coarse-Grained Distance Calculation Using Bounding Boxes --- */

//  nHit[0] = 0;
  int myHit = 0;
  double fxv[nDims*nvertf];
  float fxv_r[nDims*4];

  int nFace = (idx == 31) ? (nFiltF - 31*(nFiltF/31)) : nFiltF/31;

  int startF = (nFiltF/31) * idx;

  for (int i = 0; i < nFace; i++)
  {
    int ff = filt_faces[startF + i];

    for (int i = 0; i < 12; i++)
      fxv[i] = cutFaces[ff*stride+i];

    // Transform face points to element OOBB axes
    for (int i = 0; i < 12; i++)
      fxv_r[i] = 0.f;

    for (int k = 0; k < 4; k++)
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
          fxv_r[3*k+i] += oobb[3*i+j] * fxv[3*k+j];

    // Get OBB of face in element's axes
    float obbF[6];
    cuda_funcs::getBoundingBox<nDims,4>(fxv_r,obbF);

    cuda_funcs::getCentroid<nDims,4>(fxv,xcFf);

    float dist1 = cuda_funcs::boundingBoxDist<3>(&oobb[9], &obbF[0]);
    bool check = (dist1 <= .01f*(float)dtol);

    double dist2 = 0;
    for (int d = 0; d < 3; d++)
      dist2 += (xcFf[d] - xcC[d]) * (xcFf[d] - xcC[d]);
    dist2 = 0.5*(sqrt(dist2) + dist1);

    if (check)
    {
      // Build up list of faces with bbox intersect from the front
      myHit++;
//      atomicAdd(nHit, 1);

      int ind = min(NF1 - 1, myHit);

      if (dist2 > myDists[ind]) continue;

      myDists[ind] = dist2;
      myFaces[ind] = ff;
      while (ind > 0 && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
    else
    {
      int ind = NF1 - 1;

      if (dist2 > myDists[ind]) continue;

      myDists[ind] = dist2;
      myFaces[ind] = ff;
      while (ind > myHit && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
  }

  // Warp-reduce our sorted lists so each thread has final list of the min values
  for (int mask = warpSize/2; mask > 0; mask /= 2)
  {
    float tmpDist[32];
    int tmpFace[32];
    for (int j = 0; j < 32; j++)
    {
      tmpDist[j] = myDists[j];
      tmpFace[j] = myFaces[j];
    }

    for (int j = 0; j < 32; j++)
    {
      float d2 = __shfl_xor(tmpDist[j], mask);
      int f2 = __shfl_xor(tmpFace[j], mask);

      // Insertion sort into our list
      int ind = NF1 - 1;

      if (d2 > myDists[ind]) continue;

      myDists[ind] = d2;
      myFaces[ind] = f2;
      while (ind > 0 && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

  double myNorm[3] = {0., 0., 0.};
  double Dist = BIG_DOUBLE;
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;
  char myCorner = -1;

  // Each thread will check against 1 face

  int ff = myFaces[idx];

  if (ff >= 0)
  {
    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];
//if (PRINT) print_nodes(fxv,ff,4);
    double bboxF[6];
    cuda_funcs::getBoundingBox<3,nvertf>(fxv,bboxF);
    /// Why does this not work with shared memory (xv) in a function...?
    if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
    {
      double rst[3];
      if (mb.getRefLoc<2>(xv, bboxC, fxv, rst)) ///
        Dist = 0.;
    }

    dPoint vec;
    vec[0] = BIG_DOUBLE; vec[1] = BIG_DOUBLE; vec[2] = BIG_DOUBLE;
    if (Dist > 0.)
    {
      Dist = intersectionCheckLinear(mb, fxv, xv, bboxC, &vec[0], myCorner); /// GIVES ISSUE ON XV ('invalid __shared__ read of size 8') ON CALL TO mb.getRefLoc<2>()
      vec /= vec.norm();
    }

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (Dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      Dist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED)
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }

//    if (LINEAR)
//    {
      if (myFlag == DC_CUT) myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

      // have 'final answer' to put into global memory
      float minDist = warpAllReduceMin(Dist);
      if (Dist == minDist)
        cutFlag[ic] = myFlag;
      /// TODO
      //  if (fabs(myDist - myDist) <= .1*dtol)
      //  {
      //    // Approx. same dist. to two faces; avg. their normals to decide
      //    myDist = myDist;
      //    for (int d = 0; d < 3; d++)
      //      myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      //    nMin++;

      //    //myDot = norm*vec;
      //    double dot = 0.;
      //    for (int d = 0; d < 3; d++)
      //      dot += myNorm[d]*vec[d];

      //    if (dot < 0)
      //      myFlag = DC_HOLE; // outwards normal = inside cutting surface
      //    else
      //      myFlag = DC_NORMAL;

//      return;
//    }
  }

  dists[idx] = Dist;
  faces[idx] = ff;
  corners[idx] = myCorner;

  __syncthreads();

  // Parallel merge sort within block
  for (int width = 2; width <= blockDim.x; width *= 2)
  {
    if (idx%width == 0)
    {
      int iBegin = idx;
      int iMiddle = iBegin + width/2;
      int iEnd = idx + width;
      int i = iBegin, j = iMiddle;

      // Copy into local array for sorting out of
      for (int k = iBegin; k < iEnd; k++)
      {
        myDists[k] = dists[k];
        myFaces[k] = faces[k];
        myCorners[k] = corners[k];
      }

      __syncthreads();

      // While there are elements in the left or right runs...
      for (int k = iBegin; k < iEnd; k++)
      {
        // If left run head exists and is <= existing right run head.
        if (i < iMiddle && (j >= iEnd || myDists[i] <= myDists[j]))
        {
          dists[k] = myDists[i];
          faces[k] = myFaces[i];
          corners[k] = myCorners[i];
          i++;
        }
        else
        {
          dists[k] = myDists[j];
          faces[k] = myFaces[j];
          corners[k] = myCorners[j];
          j++;
        }
      }
//      if (PRINT) printf("idx %d: myFaces %d %d %d %d %d %d %d %d"
//                        " %d %d %d %d %d %d %d %d"
//                        " %d %d %d %d %d %d %d %d\n",
//                        idx,myFaces[0],myFaces[1],myFaces[2],myFaces[3]
//                        ,myFaces[4],myFaces[5],myFaces[6],myFaces[7]
//                        ,myFaces[8],myFaces[9],myFaces[10],myFaces[11]
//                        ,myFaces[12],myFaces[13],myFaces[14],myFaces[15]
//                        ,myFaces[16],myFaces[17],myFaces[18],myFaces[19]
//                        ,myFaces[28],myFaces[29],myFaces[30],myFaces[31]);
    }

    __syncthreads();
  }

  if (idx < NF2)
  {
    faceListOut[NF2*tid + idx] = faces[idx];
    cornerOut[NF2*tid + idx] = corners[idx];
  }
}

template<int nDims, int nSideC, int nSideF>
__global__
void cuttingPass3(dMeshBlock mb, dvec<double> cutFaces, dvec<int> checkFaces,
    int* __restrict__ cutFlag, int cutType, dvec<int> list, int ncells, dvec<char> corners)
{
  // Note: blockDim.x == 2 * 3 * nQuadFace
  const int fic = blockIdx.x;
  const int idx = threadIdx.x;

  if (fic >= ncells) return;

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;
  const int sOrderC = nSideC - 1;

  const int nQuadFace = sOrderC*sOrderC;
  const int nTriFace = 2*nQuadFace;

  const unsigned char qid = (idx % (nTriFace)) / 2;
  const unsigned char tid = (idx % (nTriFace)) % 2;

  const int ic = list[fic];  // Get filtered cell ID

  // Load up the cell nodes into shared memory

  __shared__ double xv[nDims*nvert];
  __shared__ unsigned char sflag[1];
  __shared__ float dist[7]; // For final min-reduction across block (max sOrderC = 6)

  if (idx == 0)
    sflag[0] = 0;

  for (int i = idx; i < nvert*nDims; i += blockDim.x)
  {
    int d = i % nDims;
    int v = i / nDims;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  __syncthreads();

  if (idx >= 3*nTriFace) return;

  double bboxC[2*nDims];
  double xcC[3];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv,xcC);

  /* ---- Check against our reduced list of faces ---- */

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 2e-1*href;

  float myDist = BIG_DOUBLE;
  double myNorm[3];
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;

  double fxv[nvertf*nDims];

  for (int F = 0; F < NF2; F++)
  {    
    if (myFlag == DC_CUT) continue;

    int ff = checkFaces[NF2*fic+F];
    if (ff < 0)
      continue;

    // Only checking half the element's faces; figure out which ones
    const char corner = corners[NF2*fic+F];
    const char fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

    // Get the specific sub-quadrilateral-triangle we're checking here
    const unsigned char fid = fList[idx / (nTriFace)];

    /* ---- Get our specific triangle ---- */

    double TC[9];

    // NOTE: Structured ordering  |  btm,top,left,right,front,back
    const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                         {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

    int I, J, K;
    switch (fid)
    {
      case 0: // Bottom
        I = qid / sOrderC;
        J = qid % sOrderC;
        K = 0;
        break;
      case 1: // Top
        I = qid / sOrderC;
        J = qid % sOrderC;
        K = sOrderC - 1;
        break;
      case 2: // Left
        I = 0;
        J = qid / sOrderC;
        K = qid % sOrderC;
        break;
      case 3: // Right
        I = sOrderC - 1;
        J = qid / sOrderC;
        K = qid % sOrderC;
        break;
      case 4: // Front
        I = qid / sOrderC;
        J = 0;
        K = qid % sOrderC;
        break;
      case 5: // Back
        I = qid / sOrderC;
        J = sOrderC - 1;
        K = qid % sOrderC;
        break;
    }

    int i0 = I+nSideC*(J+nSideC*K);
    int j0 = i0 + nSideC*nSideC;
    int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
    for (int i = 0; i < 8; i++)
      lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

    for (int p = 0; p < 3; p++)
    {
      int ipt = lin2curv[TriPts[fid+tid][p]];
      for (int d = 0; d < 3; d++)
        TC[3*p+d] = xv[3*ipt+d];
    }

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    // 1) In case of face entirely inside element, check if a pt is inside ele
    if (idx == 0)
    {
      double bboxF[6];
      cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

      if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
      {
        double rst[3];
        if (mb.getRefLoc<nSideC>(xv, bboxC, fxv, rst))
          sflag[0] = DC_CUT;
      }
    }

    __syncthreads();

    if (sflag[0] == DC_CUT)
    {
      if (idx == 0)
        cutFlag[ic] = (cutType != 0) ? DC_HOLE : DC_NORMAL;

      return;
    }

    // Find distance from face to cell
    dPoint vec;
    double dist = intersectionCheckOne<nSideC,nSideF>(mb, fxv, &vec[0], TC);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (dist < dtol) // They intersect 1e-8*href
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .02*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .02*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from our element; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }
  }

  if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  /* ---- Synchronize within element ---- */

  float minDist = warpAllReduceMin(myDist);

  int lane = idx % warpSize;
  int wid = idx / warpSize;

  if (lane == 0) dist[wid] = minDist;

  __syncthreads();

  minDist = (idx < blockDim.x / warpSize) ? dist[lane] : 0;

  if (wid == 0)
  {
    minDist = warpAllReduceMin(minDist);
    dist[0] = minDist;
  }

  __syncthreads();

  minDist = dist[0];

  // Thread with minimum ('best') distance sets final cutting flag
  // Race condition if multiple threads have dist 0, but then all will be
  // setting the same value ('DC_CUT') anyways
  if (myDist == minDist)
    cutFlag[ic] = myFlag;
}

/*! Remove all elements which do not intersect with cut group's bbox from
 *  consideration (obviously do not intersect) */
template<int nvert>
__global__
void filterElements(dMeshBlock mb, dvec<double> cut_bbox, dvec<int> filt,
    dvec<int> cutFlag, dvec<int> nfilt, dvec<float> bboxOut)
{
  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ic == 0)
  {
    nfilt[0] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1.e10f;
      bboxOut[d+3] = -1.e10f;
    }
  }

  __shared__ float bboxF[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxF[i] = (float)cut_bbox[i];

  __syncthreads();

  if (ic >= mb.ncells) return;

  // Set all cell flags initially to DC_NORMAL (filtered cells will remain 'NORMAL')
  cutFlag[ic] = DC_NORMAL;

  float href = .005/3.*(bboxF[3]-bboxF[0]+bboxF[4]-bboxF[1]+bboxF[5]-bboxF[2]);

  // Get element nodes
  float xv[nvert*3];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = (float)mb.coord[ic+mb.ncells*(d+3*i)];

  // Get element bounding box
  float bboxC[6], xc[3];
  cuda_funcs::getBoundingBox<3,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<3,nvert>(xv, xc);

  if (mb.rrot) // Transform xc to hole map's coordinate system
  {
    double x2[3] = {0.,0.,0.};
    for (int d1 = 0; d1 < 3; d1++)
      for (int d2 = 0; d2 < 3; d2++)
        x2[d1] += mb.Rmat[d1+3*d2]*(xc[d2]-mb.offset[d2]);

    for (int d = 0; d < 3; d++)
      xc[d] = x2[d];
  }

  char tag = cuda_funcs::checkHoleMap(xc, mb.hm_sam.data(), mb.hm_nx.data(), mb.hm_extents.data());
  bool checkH = (tag != 1);
  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, href);

  // If filtering element due to being completely inside hole region, tag as hole
  if (tag == 1)
    cutFlag[ic] = DC_HOLE;

  if (checkH && checkB)
  {
    int ind = atomicAggInc(&nfilt[0]);
    filt[ind] = ic;
    for (int d = 0; d < 3; d++)
    {
      atomicMinf(&bboxOut[d], bboxC[d]);
      atomicMaxf(&bboxOut[d+3], bboxC[d+3]);
    }
  }
}

/*! Remove all cutting faces which do not intersect this rank's reduced bbox
 *  from consideration (obviously do not intersect) */
template<int nvertf>
__global__
void filterFaces(dMeshBlock mb, dvec<float> ele_bbox, int nCut,
    dvec<double> cutFaces, dvec<int> filt, dvec<int> nfilt, dvec<float> bboxOut)
{
  const unsigned int ff = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ff == 0)
  {
    nfilt[1] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1e10f;
      bboxOut[d+3] = -1e10f;
    }
  }

  __shared__ float bboxE[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxE[i] = ele_bbox[i];

  __syncthreads();

  if (ff >= nCut) return;

  float href = .01f/3.f*(bboxE[3]-bboxE[0]+bboxE[4]-bboxE[1]+bboxE[5]-bboxE[2]);

  // Get face nodes
  float fxv[nvertf*3];
  for (int i = 0; i < nvertf; i++)
    for (int d = 0; d < 3; d++)
      fxv[3*i+d] = (float)cutFaces[(ff*nvertf+i)*3+d];

  // Get face bounding box
  float bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

  /// TODO: apply Rmat, offset to xc!
  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxF, bboxE, href);

  if (checkB)
  {
    int ind = atomicAggInc(&nfilt[1]);
    filt[ind] = ff;
    for (int d = 0; d < 3; d++)
    {
       atomicMinf(&bboxOut[d], bboxF[d]);
       atomicMaxf(&bboxOut[d+3], bboxF[d+3]);
    }
  }
}

void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, double *cutBbox_h, int* cutFlag, int cutType)
{
  // Setup cutMap TODO: create initialization elsewhere?
  cutFlag_d.resize(ncells);
  filt_eles.resize(ncells);
  filt_faces.resize(nCut);

  dvec<double> cutFaces;
  cutFaces.assign(cutFaces_h, nCut*nvertf*nDims);

  dvec<double> cutBbox_d;
  cutBbox_d.assign(cutBbox_h, 2*nDims);
  if (nDims != 3) printf("Bad nDims!!!! nDims = %d\n",nDims); /// DEBUGGING

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);
  }

  // Filter elements based upon cutting-surface bounding box

  hvec<int> nfilt_h;
  dvec<int> nfilt_d;
  nfilt_h.resize(2);
  nfilt_h[0] = 0;  nfilt_h[1] = 0;
  nfilt_d.assign(nfilt_h.data(), nfilt_h.size());

  ele_bbox.resize(6);
  face_bbox.resize(6);

  int threads = 128;
  int blocks = (ncells + threads - 1) / threads;

  switch(nvert)
  {
    case 8:
      filterElements<8><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox);
      break;
//    case 27:
//      filterElements<27><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
    case 64:
      filterElements<64><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox);
      break;
//    case 125:
//      filterElements<125><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
    default:
      printf("nvert = %d\n",nvert);
      ThrowException("nvert case not implemented for filterElements on device");
  }

  check_error();

  if (nCut == 0)
  {
    cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

    cutFaces.free_data();
    cutBbox_d.free_data();
    nfilt_h.free_data();
    nfilt_d.free_data();

    return;
  }

  blocks = (nCut + threads - 1) / threads;

  switch(nvertf)
  {
    case 4:
      filterFaces<4><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    case 16:
      filterFaces<16><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    default:
      printf("nvertf = %d\n",nvertf);
      ThrowException("nvertf case not implemented for filterFaces on device");
  }

  hipDeviceSynchronize();
  check_error();

  nfilt_h.assign(nfilt_d.data(), 2);
  int nfiltC = nfilt_h[0];
  int nfiltF = nfilt_h[1];
  printf("nfilt = %d, %d\n",nfilt_h[0], nfilt_h[1]);

  // Perform the Direct Cut algorithm on the filtered list of grid elements

  dvec<int> checkFaces;
  dvec<char> corners;
  checkFaces.resize(NF2*nfiltC);
  corners.resize(NF2*nfiltC);

  threads = 32;
  blocks = (nfiltC + threads - 1) / threads;
  int nbShare = sizeof(double)*4*nDims;

/*
//  if (nfiltC > 0)
//  {
//    switch(nvertf)
//    {
//      case 4:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, nfilt_h[1], filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces);
//            break;
////          case 27:
////            fillCutMap<3,3,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 64:
////            fillCutMap<3,4,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 125:
////            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          default:
////            printf("nvert = %d\n",nvert);
////            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
////      case 9:
////        switch(nvert)
////        {
////          case 8:
////            fillCutMap<3,2,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 27:
////            fillCutMap<3,3,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 64:
////            fillCutMap<3,4,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 125:
////            fillCutMap<3,5,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          default:
////            printf("nvert = %d\n",nvert);
////            ThrowException("nvert case not implemented for directCut on device");
////        }
////        break;
//      case 16:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, nfilt_h[1], filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces);
//            break;
////          case 27:
////            fillCutMap<3,3,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 64:
////            fillCutMap<3,4,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 125:
////            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
////      case 25:
////        switch(nvert)
////        {
////          case 8:
////            fillCutMap<3,2,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 27:
////            fillCutMap<3,3,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 64:
////            fillCutMap<3,4,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          case 125:
////            fillCutMap<3,5,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
////            break;
////          default:
////            printf("nvert = %d\n",nvert);
////            ThrowException("nvert case not implemented for directCut on device");
////        }
////        break;
//      default:
//        printf("nvertFace = %d\n",nvertf);
//        ThrowException("nvertFace case not implemented for directCut on device");
//    }
//  }
*/

  if (nfiltC > 0)
  {
    switch(nvertf)
    {
//      case 4:
//        switch(nvert)
//        {
//          case 8:
//          {
//            blocks = nfiltC;
//            threads = 32;
//            nbShare = (nvert*nDims)*sizeof(double) + 32*sizeof(float) + 32*sizeof(int) + 1*sizeof(int) + 32*sizeof(char);

//            cuttingPass1<3,2,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nfiltF, filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces, corners);

//            hipDeviceSynchronize();
//            check_error();

//            /* ---- Pass 3: 'Finest-Grained' Direct Cut Check ---- */

//            int nSideC = std::cbrt(nvert);
//            int nTri = 3*2*(nSideC-1)*(nSideC-1);
//            int nbshared = nDims*nvert*sizeof(double) + 7*sizeof(float) + 1;

//            cuttingPass3<3,2,2><<<nfiltC, nTri, nbshared>>>(*this, cutFaces, checkFaces, cutFlag_d.data(), cutType, eleList, nfiltC, corners);

//            hipDeviceSynchronize();
//            check_error();

//            break;
//          }
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;

      case 16:
        switch(nvert)
        {
          case 8:
          {
            hipDeviceSynchronize();
            check_error();

            blocks = nfiltC;
            threads = 32;
            nbShare = (nvert*nDims)*sizeof(double) + 32*sizeof(float) + (32+1)*sizeof(int) + 32*sizeof(char);

            cuttingPass1<3,2,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nfiltF, filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces, corners);

            hipDeviceSynchronize();
            check_error();

            /* ---- Pass 3: 'Finest-Grained' Direct Cut Check ---- */

            int nSideC = std::cbrt(nvert);
            int nTri = 3*2*(nSideC-1)*(nSideC-1);
            int nbshared = nDims*nvert*sizeof(double) + 7*sizeof(float) + 1;

            cuttingPass3<3,2,4><<<blocks, nTri, nbshared>>>(*this, cutFaces, checkFaces, cutFlag_d.data(), cutType, filt_eles, nfiltC, corners);

            hipDeviceSynchronize();
            check_error();

            break;
          }
          case 64:
          {
            hipDeviceSynchronize();
            check_error();

            blocks = nfiltC;
            threads = 32;
            nbShare = (nvert*nDims)*sizeof(double) + 32*sizeof(float) + (32+1)*sizeof(int) + 32*sizeof(char);

            cuttingPass1<3,4,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nfiltF, filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces, corners);

            hipDeviceSynchronize();
            check_error();

            /* ---- Pass 3: 'Finest-Grained' Direct Cut Check ---- */

            int nSideC = std::cbrt(nvert);
            int nTri = 3*2*(nSideC-1)*(nSideC-1);
            int nbshared = nDims*nvert*sizeof(double) + 7*sizeof(float) + 1;

            cuttingPass3<3,4,4><<<blocks, nTri, nbshared>>>(*this, cutFaces, checkFaces, cutFlag_d.data(), cutType, filt_eles, nfiltC, corners);

            hipDeviceSynchronize();
            check_error();

            break;
          }
          default:
            printf("nvert = %d\n",nvert);
            ThrowException("nvert case not implemented for directCut on device");
        }
        break;
      default:
        printf("nvertFace = %d, nCut = %d\n",nvertf,nCut);
        ThrowException("nvertFace case not implemented for directCut on device");
    }
  }

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  nfilt_d.free_data();
  nfilt_h.free_data();

  corners.free_data();
  checkFaces.free_data();

  cutFaces.free_data();
  cutBbox_d.free_data();
}
