#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

#include "hip/device_functions.h"
#include "math.h"

#define MAX_UCHAR 255

/* --- Handy Vector Operation Macros --- */

#define NF1 32 // 20-32 depending on unstructured-ness of grid & desire for robustness
#define NF2  4 // 3-6 depending on unstructured-ness of grid & desire for robustness

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
double DOTCROSS4(const double* __restrict__ c,
                 const double* __restrict__ a1, const double* __restrict__ a2,
                 const double* __restrict__ b1, const double* __restrict__ b2)
{
  double d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}

static
__device__ __forceinline__
float DOTCROSS4(const float* __restrict__ c,
                 const float* __restrict__ a1, const float* __restrict__ a2,
                 const float* __restrict__ b1, const float* __restrict__ b2)
{
  float d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}


/* --- Misc. Helpful CUDA kernels --- */

__device__ __forceinline__
void print_nodes(const double* pts, int id, int npts)
{
  int idx = threadIdx.x;
  for (int tid = 0; tid < 32; tid++)
  {
    if (idx == tid)
    {
      printf("Points%d = [",id);
      for (int i = 0; i < npts - 1; i++)
        printf("%f %f %f;\n",pts[3*i+0],pts[3*i+1],pts[3*i+2]);

      int I = npts-1;
      printf("%f %f %f];\n",pts[3*I+0],pts[3*I+1],pts[3*I+2]);
    }
  }
}

#define WARP_SZ 32

__device__
inline int lane_id(void) { return threadIdx.x % WARP_SZ; }

__device__
inline int warp_bcast(int v, int leader) { return __shfl(v, leader); }

__device__ __forceinline__
float warpAllReduceMin(float val)
{
  for (int mask = warpSize/2; mask > 0; mask /= 2)
    val = fminf(val, __shfl_xor(val, mask));
  return val;
}

__device__
int floatToOrderedInt(float floatVal)
{
  int intVal = __float_as_int(floatVal);

  return (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF;
}

__device__
unsigned int floatToUint(float fval)
{
  unsigned int ival = __float_as_uint(fval);
  unsigned int mask = -int(ival >> 31) | 0x80000000;
  return ival ^ mask;
}

__device__
float uintToFloat(unsigned int ival)
{
  unsigned int mask = ((ival >> 31) - 1) | 0x80000000;
  return __uint_as_float(ival ^ mask);
}

__device__
float orderedIntToFloat(int intVal)
{
  return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF );
}

__device__ float atomicMaxf(float* address, float val)
{
  //int *iaddr = (int*)address;
  int old = __float_as_int(*address);
  int assumed;
  while (val > __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

__device__ float atomicMinf(float* address, float val)
{
  int old = __float_as_int(*address);
  int assumed;
  while (val < __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

/*! Warp-aggregated atomic increment
 *  https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/ */
__device__
int atomicAggInc(int *ctr)
{
  int mask = __ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  int res;
  if (lane_id() == leader)
    res = atomicAdd(ctr, __popc(mask));
  // brodcast result
  res = warp_bcast(res, leader);
  // each thread computes its own value
  return res + __popc(mask & ((1 << lane_id()) - 1));
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch, int rank)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->rank = rank;

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  std::vector<float> xlistf_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  for (int i = 0; i < nSide; i++)
    xlistf_h[i] = xlist_h[i];

  xlist.assign(xlist_h.data(), xlist_h.size());
  xlistf.assign(xlistf_h.data(), xlistf_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::assignHoleMap(bool hasWall, int* nx, int* sam, double* extents)
{
  if (hasWall)
  {
    int size = nx[0]*nx[1]*nx[2];

    std::vector<char> tmp_sam(size);
    for (int i = 0; i < size; i++)
      tmp_sam[i] = (char)sam[i];

    double dx[3];
    for (int d = 0; d < 3; d++)
      dx[d] = (extents[d+3] - extents[d]) / nx[d];

    hm_sam.assign(tmp_sam.data(), size);
    hm_extents.assign(extents, 6);
    hm_nx.assign(nx, 3);
    hm_dx.assign(dx, 3);
  }
  else
  {
    clearHoleMap();
  }
}

void dMeshBlock::clearHoleMap(void)
{
  int nx[3] = {0,0,0};
  double dx[3] = {0,0,0};
  double extents[6] = {0,0,0,0,0,0};

  hm_sam.resize(0);

  hm_nx.assign(nx, 3);
  hm_dx.assign(dx, 3);
  hm_extents.assign(extents, 6);
}

void dMeshBlock::updateADTData(int ncells_adt, int* eleList, double* eleBBox)
{
  this->eleBBox.assign(eleBBox, ncells_adt*nDims*2);
  this->eleList.assign(eleList, ncells_adt);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true; /// WORKING ON ADT REBUILD - DISABLED RROT
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
double lineSegmentDistance(double *p1, double *p2, double *p3, double *p4, double *dx)
{
  // Get the line equations
  double U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  double V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  double W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  double uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  double vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  double uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  double uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  double vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  double den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  double s = (den < 1e-10) ? 0 : (uv*vw - vv*uw) / den;
  double t = (den < 1e-10) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.), 1.);
  t = fmin(fmax(t, 0.), 1.);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  double dist = 0;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

static
__device__
float lineSegmentDistance(float *p1, float *p2, float *p3, float *p4, float *dx)
{
  // Get the line equations
  const float U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  const float V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  const float W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  const float uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  const float vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  const float uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  const float uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  const float vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  const float den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  float s = (den < 1e-10f) ? 0.0f : (uv*vw - vv*uw) / den;
  float t = (den < 1e-10f) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.f), 1.f);
  t = fmin(fmax(t, 0.f), 1.f);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  float dist = 0.f;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise 
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
double triTriDistance2(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const double* V01 = T1;
  const double* V11 = T1+3;
  const double* V21 = T1+6;

  const double* V02 = T2;
  const double* V12 = T2+3;
  const double* V22 = T2+6;

  double N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  double norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  double d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  double d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  double d01 = DOT(N2,V01) + d2;
  double d11 = DOT(N2,V11) + d2;
  double d21 = DOT(N2,V21) + d2;

  double d02 = DOT(N1,V02) + d1;
  double d12 = DOT(N1,V12) + d1;
  double d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      double P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      double P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      double P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      double P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      double P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      double P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  double L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  double p0 = DOT(L,V01);
  double p1 = DOT(L,V11);
  double p2 = DOT(L,V21);

  double q0 = DOT(L,V02);
  double q1 = DOT(L,V12);
  double q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
float triTriDistance2(float* T1, float* T2, float* minVec, float tol)
{
  float dist = 1e15f;
  float vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      float D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const float* V01 = T1;
  const float* V11 = T1+3;
  const float* V21 = T1+6;

  const float* V02 = T2;
  const float* V12 = T2+3;
  const float* V22 = T2+6;

  float N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  float norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  float d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  float d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  float d01 = DOT(N2,V01) + d2;
  float d11 = DOT(N2,V11) + d2;
  float d21 = DOT(N2,V21) + d2;

  float d02 = DOT(N1,V02) + d1;
  float d12 = DOT(N1,V12) + d1;
  float d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      float P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      float P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      float P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      float P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      float P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      float P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  float L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  float p0 = DOT(L,V01);
  float p1 = DOT(L,V11);
  float p2 = DOT(L,V21);

  float q0 = DOT(L,V02);
  float q1 = DOT(L,V12);
  float q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  float s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  float t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10f) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10f) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10f) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10f) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    float dt = fmin(fabs(t1-s2), fabs(s1-t2));
    float dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.f;
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
float triTriDistance3(float* T1, float* T2, float tol)
{
  float dist = 1e15f;
  float vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      float D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        dist = D;
      }
    }
  }

  // Pointers to points
  const float* V01 = T1;
  const float* V11 = T1+3;
  const float* V21 = T1+6;

  const float* V02 = T2;
  const float* V12 = T2+3;
  const float* V22 = T2+6;

  float N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  float norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  float d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  float d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  float d01 = DOT(N2,V01) + d2;
  float d11 = DOT(N2,V11) + d2;
  float d21 = DOT(N2,V21) + d2;

  float d02 = DOT(N1,V02) + d1;
  float d12 = DOT(N1,V12) + d1;
  float d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      float P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      float P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      float P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      float P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      float P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      float P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  float L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  float p0 = DOT(L,V01);
  float p1 = DOT(L,V11);
  float p2 = DOT(L,V21);

  float q0 = DOT(L,V02);
  float q1 = DOT(L,V12);
  float q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  float s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  float t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10f) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10f) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10f) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10f) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    float dt = fmin(fabs(t1-s2), fabs(s1-t2));
    float dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
      dist = dl;

    return dist;
  }

  return 0.f;
}


static
__device__ __forceinline__
dPoint faceNormal(const double* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPoint pt0 = dPoint(&xv[0]);
  dPoint pt1 = dPoint(&xv[3]);
  dPoint pt2 = dPoint(&xv[6]);
  dPoint norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPoint(&xv[9]);
  dPoint norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPoint norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

static
__device__ __forceinline__
dPointf faceNormal(const float* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPointf pt0 = dPointf(&xv[0]);
  dPointf pt1 = dPointf(&xv[3]);
  dPointf pt2 = dPointf(&xv[6]);
  dPointf norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPointf(&xv[9]);
  dPointf norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPointf norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

template<int nSideC, int nSideF>
__device__
double intersectionCheckOne(dMeshBlock &mb, const double* __restrict__ fxv,
    double* __restrict__ minVec, double* TC)
{
  /* --- Prerequisites --- */

  const int sorderF = nSideF-1;

  double tol = 1e-9;
  double TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  for (int M = 0; M < sorderF; M++)
  {
    for (int N = 0; N < sorderF; N++)
    {
      int m0 = M + nSideF*N;
      int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
      for (int m = 0; m < 2; m++)
        for (int n = 0; n < 3; n++)
          TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

template<int nSideC, int nSideF>
__device__
float intersectionCheckOne(dMeshBlock &mb, const float* __restrict__ fxv,
    float* __restrict__ minVec, float* TC)
{
  /* --- Prerequisites --- */

  const int sorderF = nSideF-1;

  float tol = 1e-9;
  float TF[9];
  float minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  for (int M = 0; M < sorderF; M++)
  {
    for (int N = 0; N < sorderF; N++)
    {
      int m0 = M + nSideF*N;
      int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
      for (int m = 0; m < 2; m++)
        for (int n = 0; n < 3; n++)
          TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        float vec[3];
        float dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}


template<int nSideC, int nSideF>
__device__
double intersectionCheck(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec)
{
  /* --- Prerequisites --- */

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  const int sorderC = nSideC-1;
  const int sorderF = nSideF-1;

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                       {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  double tol = 1e-9;
  double TC[9], TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,nvert>(exv, bboxC);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<nSideC>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Check outer faces of element for intersection with face
#pragma unroll
  for (int f = 0; f < 6; f++)
  {
#pragma unroll
    for (int g = 0; g < sorderC*sorderC; g++)
    {
      int I, J, K;
      switch (f)
      {
        case 0: // Bottom
          I = g / sorderC;
          J = g % sorderC;
          K = 0;
          break;
        case 1: // Top
          I = g / sorderC;
          J = g % sorderC;
          K = sorderC - 1;
          break;
        case 2: // Left
          I = 0;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 3: // Right
          I = sorderC - 1;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 4: // Front
          I = g / sorderC;
          J = 0;
          K = g % sorderC;
          break;
        case 5: // Back
          I = g / sorderC;
          J = sorderC - 1;
          K = g % sorderC;
          break;
      }

      int i0 = I+nSideC*(J+nSideC*K);
      int j0 = i0 + nSideC*nSideC;
      int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
      for (int i = 0; i < 8; i++)
        lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

      // Get triangles for the sub-hex of the larger curved hex
      for (int i = f; i < f+2; i++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = lin2curv[TriPts[i][p]];
          for (int d = 0; d < 3; d++)
            TC[3*p+d] = exv[3*ipt+d];
        }

        cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
        double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
        btol = fmin(btol, minDist);
        if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;

        for (int M = 0; M < sorderF; M++)
        {
          for (int N = 0; N < sorderF; N++)
          {
            int m0 = M + nSideF*N;
            int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
            for (int m = 0; m < 2; m++)
              for (int n = 0; n < 3; n++)
                TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

            // Intersection check between element face tris & cutting-face tris
            for (int j = 0; j < 2; j++)
            {
              for (int p = 0; p < 3; p++)
              {
                int ipt = TriPtsF[j][p];
                for (int d = 0; d < 3; d++)
                  TF[3*p+d] = fxv[3*ipt+d];
              }

              double vec[3];
              double dist = triTriDistance2(TF, TC, vec, tol);

              if (dist < tol)
                return 0.;

              if (dist < minDist)
              {
                for (int d = 0; d < 3; d++)
                  minVec[d] = vec[d];
                minDist = dist;
              }
            }
          }
        }
      }
    }
  }

  // 3) Definitely no intersection; use centroids to get vector
  if (minDist == BIG_DOUBLE)
  {
    double tmp[3];
    cuda_funcs::getCentroid<3,nvert>(exv,minVec);
    cuda_funcs::getCentroid<3,nvertf>(fxv,tmp);

    minDist = 0;
    for (int d = 0; d < 3; d++) // Vector is face -> cell
    {
      minVec[d] -= tmp[d];
      minDist += minVec[d]*minVec[d];
    }

    return sqrt(minDist);
  }

  return minDist;
}

__device__ __forceinline__
double intersectionCheckLinear(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, const double* __restrict__ bboxC,
    double* __restrict__ minVec, char &cornerOut)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  double tol = 1e-8f;
  double TC[9], TF[9];
  double minDist = 1e15;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxF[6];
  cuda_funcs::getBoundingBox<3,4>(fxv, bboxF);

  double xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element (Checked outside this func)
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 2) Find nearest corner of element to face; check only that half of element

  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    double dist = 0.;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        minVec[d] = exv[3*i+d] - xcf[d];
      corner = i;
    }
  }

  cornerOut = corner;

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const int fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

  // 3) Check those faces of element for intersection with face
#pragma unroll
  for (int F = 0; F < 3; F++)
  {
    int f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        int ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      const int TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

__device__ __forceinline__
float intersectionCheckLinear(dMeshBlock &mb, const float* __restrict__ fxv,
    const float* __restrict__ exv, const float* __restrict__ bboxC,
    float* __restrict__ minVec, char &cornerOut)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  float tol = 1e-8f;
  float TC[9], TF[9];
  float minDist = 1e15f;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  float xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element (Checked outside this func)
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 2) Find nearest corner of element to face; check only that half of element

  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    float dist = 0.f;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        minVec[d] = exv[3*i+d] - xcf[d];
      corner = i;
    }
  }

  cornerOut = corner;

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const int fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

  // 3) Check those faces of element for intersection with face
#pragma unroll
  for (int F = 0; F < 3; F++)
  {
    int f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        int ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      const int TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        float vec[3];
        float dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

__device__ __forceinline__
float intersectionCheckLinear2(const float* __restrict__ fxv,
    const float* __restrict__ exv, char &cornerOut)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  float tol = 1e-7f;
  float TC[9], TF[9];
  float minDist = 1e15f;

  float xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  // Find nearest corner of element to face; check only that half of element
  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    float dist = 0.f;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      corner = i;
    }
  }

  cornerOut = corner;

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const char fList[3] = {(char)(corner / 4), (char) (((corner + 1)%4) / 2 + 2), (char)(((corner%4) / 2) + 4)};

  // 3) Check those faces of element for intersection with face
  for (int F = 0; F < 3; F++)
  {
    char f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        char ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      const char TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          char ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        float dist = triTriDistance3(TF, TC, tol);

        if (dist < tol)
          return 0.;

        dist = (minDist < dist) ? minDist : dist;
      }
    }
  }

  return minDist;
}

__global__
void cuttingPass1(dMeshBlock mb, dvec<float> cutFaces, int nvertf, int nFiltF,
    dvec<int> filt_faces, int* __restrict__ cutFlag, int cutType,
    int* __restrict__ list, int ncells, dvec<int> faceListOut, dvec<char> cornerOut)
{
  const int tid = blockIdx.x;
  const int idx = threadIdx.x;
  /// USING 32 THREADS PER BLOCK - 1 CELL PER BLOCK

  if (tid >= ncells) return;

  const int nDims = 3;

  const int ic = list[tid];  // Get filtered cell ID
  const int stride = nDims*nvertf;
  const bool LINEAR = (nvertf == 4);

  //bool PRINT = threadIdx.x == 0 && blockIdx.x == 0;

  __shared__ float xv[nDims*8];
  __shared__ float dists[32];
  __shared__ int faces[32];
  __shared__ int nHit[1];
  __shared__ char corners[32];

  __syncthreads();

  // Load up the cell nodes into an array
  for (int i = idx; i < 8*nDims; i += blockDim.x)
  {
    int d = i % 3;
    int v = i / 3;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  __syncthreads();

  int myFaces[32];  
  float myDists[32];
  char myCorners[32];

  for (int i = 0; i < 32; i++)
  {
    myFaces[i] = -1;
    myDists[i] = BIG_FLOAT;
    myCorners[i] = -1;
  }

  float bboxC[2*nDims];
  float xcC[3];
  float oobb[15];
  cuda_funcs::getBoundingBox<nDims,8>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,8>(xv, xcC);
  cuda_funcs::getOOBB<nDims,8>(xv,oobb);

  const float href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const float dtol = 2e-1f*href;

  /* --- Pass 1: Coarse-Grained Distance Calculation Using Bounding Boxes --- */

  nHit[0] = 0;
  int myHit = 0;
  float fxv[nDims*4];

  int nFace = (idx == 31) ? (nFiltF - 31*(nFiltF/31)) : nFiltF/31;

  int startF = (nFiltF/31) * idx;

  for (int i = 0; i < nFace; i++)
  {
    int ff = filt_faces[startF + i];

    for (int i = 0; i < nDims*4; i++)
      fxv[i] = cutFaces[ff*stride+i];

    // Transform face points to element OOBB axes
    float fxv_r[nDims*4];
    for (int i = 0; i < 12; i++)
      fxv_r[i] = 0.f;

    for (int k = 0; k < 4; k++)
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
          fxv_r[3*k+i] += oobb[3*i+j] * fxv[3*k+j];

    // Get OBB of face in element's axes
    float obbF[6];
    cuda_funcs::getBoundingBox<nDims,4>(fxv_r,obbF);

    float xcFf[3];
    cuda_funcs::getCentroid<nDims,4>(fxv,xcFf);

    float dist1 = cuda_funcs::boundingBoxDist<3>(&oobb[9], &obbF[0]);
    bool check = (dist1 <= .01f*dtol);

    float dist2 = 0.f;
    for (int d = 0; d < 3; d++)
      dist2 += (xcFf[d] - xcC[d]) * (xcFf[d] - xcC[d]);
    dist2 = 0.5f*(sqrt(dist2) + dist1);

    if (check)
    {
      // Build up list of faces with bbox intersect from the front
      myHit++;
//      atomicAdd(nHit, 1);

      int ind = min(NF1 - 1, myHit);

      if (dist2 > myDists[ind]) continue;

      myDists[ind] = dist2;
      myFaces[ind] = ff;
      while (ind > 0 && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
    else
    {
      int ind = NF1 - 1;

      if (dist2 > myDists[ind]) continue;

      myDists[ind] = dist2;
      myFaces[ind] = ff;
      while (ind > myHit && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
  }

  // Warp-reduce our sorted lists so each thread has final list of the min values
  for (int mask = warpSize/2; mask > 0; mask /= 2)
  {
    float tmpDist[32];
    int tmpFace[32];
    for (int j = 0; j < 32; j++)
    {
      tmpDist[j] = myDists[j];
      tmpFace[j] = myFaces[j];
    }

    for (int j = 0; j < 32; j++)
    {
      float d2 = __shfl_xor(tmpDist[j], mask);
      int f2 = __shfl_xor(tmpFace[j], mask);

      // Insertion sort into our list
      int ind = NF1 - 1;

      if (d2 > myDists[ind]) continue;

      myDists[ind] = d2;
      myFaces[ind] = f2;
      while (ind > 0 && myDists[ind] < myDists[ind-1])
      {
        swap(myDists[ind], myDists[ind-1]);
        swap(myFaces[ind], myFaces[ind-1]);
        ind--;
      }
    }
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

//  float myNorm[3] = {0., 0., 0.};
  float Dist = BIG_FLOAT;
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;
  char myCorner = -1;

  // Each thread will check against 1 face

  int ff = myFaces[idx];

  if (ff >= 0)
  {
    // Load face vertices
    for (int i = 0; i < 4*nDims; i++)
      fxv[i] = cutFaces[ff*stride+i];

    float bboxF[6];
    cuda_funcs::getBoundingBox<3,4>(fxv,bboxF);
    /// Why does this not work with shared memory (xv) in a function...?
    if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
    {
      float rst[3];
      if (mb.getRefLoc<2>(xv, bboxC, fxv, rst)) ///
        Dist = 0.;
    }

    dPointf vec;
    vec[0] = BIG_DOUBLE; vec[1] = BIG_DOUBLE; vec[2] = BIG_DOUBLE;
    if (Dist > 0.)
    {
      Dist = intersectionCheckLinear(mb, fxv, xv, bboxC, &vec[0], myCorner); /// GIVES ISSUE ON XV ('invalid __shared__ read of size 8') ON CALL TO mb.getRefLoc<2>()
      vec /= vec.norm();
    }

    dPointf norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (Dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      Dist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED)
    {
      // Unflagged cell, or have a closer face to use
      float dot = norm*vec;

//      for (int d = 0; d < 3; d++)
//        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }

    if (LINEAR)
    {
      if (myFlag == DC_CUT) myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

      // have 'final answer' to put into global memory
      float minDist = warpAllReduceMin(Dist);
      if (Dist == minDist)
        cutFlag[ic] = myFlag;
      /// TODO
      //  if (fabs(myDist - myDist) <= .1*dtol)
      //  {
      //    // Approx. same dist. to two faces; avg. their normals to decide
      //    myDist = myDist;
      //    for (int d = 0; d < 3; d++)
      //      myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      //    nMin++;

      //    //myDot = norm*vec;
      //    double dot = 0.;
      //    for (int d = 0; d < 3; d++)
      //      dot += myNorm[d]*vec[d];

      //    if (dot < 0)
      //      myFlag = DC_HOLE; // outwards normal = inside cutting surface
      //    else
      //      myFlag = DC_NORMAL;

      return;
    }
  }

  dists[idx] = Dist;
  faces[idx] = ff;
  corners[idx] = myCorner;

  __syncthreads();

  // Parallel merge sort within block
  for (int width = 2; width <= blockDim.x; width *= 2)
  {
    if (idx%width == 0)
    {
      int iBegin = idx;
      int iMiddle = iBegin + width/2;
      int iEnd = idx + width;
      int i = iBegin, j = iMiddle;

      // Copy into local array for sorting out of
      for (int k = iBegin; k < iEnd; k++)
      {
        myDists[k] = dists[k];
        myFaces[k] = faces[k];
        myCorners[k] = corners[k];
      }

      __syncthreads();

      // While there are elements in the left or right runs...
      for (int k = iBegin; k < iEnd; k++)
      {
        // If left run head exists and is <= existing right run head.
        if (i < iMiddle && (j >= iEnd || myDists[i] <= myDists[j]))
        {
          dists[k] = myDists[i];
          faces[k] = myFaces[i];
          corners[k] = myCorners[i];
          i++;
        }
        else
        {
          dists[k] = myDists[j];
          faces[k] = myFaces[j];
          corners[k] = myCorners[j];
          j++;
        }
      }
    }

    __syncthreads();
  }

  if (idx < NF2)
  {
    faceListOut[NF2*tid + idx] = faces[idx];
    cornerOut[NF2*tid + idx] = corners[idx];
  }
}


__global__
void cuttingPass1New(dMeshBlock mb, dvec<int> filt_eles, int nEles,
    dvec<float> cutFaces, int nvertf, int nFaces, dvec<int> filt_faces,
    dvec<char> outCorner, dvec<float> outDist)
{
  const int IC = blockIdx.x * blockDim.x + threadIdx.x;
  const int F = blockIdx.y * blockDim.y + threadIdx.y;

  if (IC >= nEles || F >= nFaces) return;

  const int nDims = 3;

  const int ic = filt_eles[IC];  // Get filtered cell ID
  const int ff = filt_faces[F];
  const int stride = nDims*nvertf;
//  const bool LINEAR = (nvertf == 4); /// TODO...?

  // Load up the cell nodes into an array
  float xv[8*nDims];
  for (int i = 0; i < 8*nDims; i++)
  {
    int d = i % 3;
    int v = i / 3;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

  // Each thread will check against 1 face
  outDist[nEles*F+IC] = intersectionCheckLinear2(&cutFaces[ff*stride], xv, outCorner[nEles*F+IC]);
}

__global__
void sortFaces(dvec<int> faceList, int nEles, int nFaces, dvec<float> distList,
    dvec<char> cornerList, dvec<int> outFaces, dvec<char> outCorners)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;

  if (IC >= nEles) return;

  float dists[NF2];
  int faces[NF2];
  char corners[NF2];

  for (int i = 0; i < NF2; i++)
  {
    dists[i] = BIG_FLOAT;
    faces[i] = -1;
    corners[i] = -1;
  }

  for (int F = 0; F < nFaces; F++)
  {
    int ind = NF2-1;

    float Dist = distList[nEles*F+IC];
    if (Dist > dists[ind]) continue;

    dists[ind] = Dist;
    faces[ind] = faceList[F];
    corners[ind] = cornerList[nEles*F+IC];
    while (ind > 0 && dists[ind] < dists[ind-1])
    {
      swap(dists[ind], dists[ind-1]);
      swap(faces[ind], faces[ind-1]);
      swap(corners[ind], corners[ind-1]);
      ind--;
    }
  }

  for (int i = 0; i < NF2; i++)
  {
    outFaces[nEles*i+IC] = faces[i];
    outCorners[nEles*i+IC] = corners[i];
  }
}

template<int nSideC, int nSideF>
__global__
void cuttingPass3(dMeshBlock mb, dvec<float> cutFaces, dvec<int> checkFaces,
    int* __restrict__ cutFlag, int cutType, dvec<int> list, int ncells, dvec<char> corners)
{
  // Note: blockDim.x == 2 * 3 * nQuadFace
  const int fic = blockIdx.x;
  const int idx = threadIdx.x;
/// TODO: have each thread be a single triangle + face check combo [use global-mem atomics for final element dist/flag]
  if (fic >= ncells) return;

  const int nDims = 3;
  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;
  const int sOrderC = nSideC - 1;

  const int nQuadFace = sOrderC*sOrderC;
  const int nTriFace = 2*nQuadFace;

  const unsigned char qid = (idx % (nTriFace)) / 2;
  const unsigned char tid = (idx % (nTriFace)) % 2;

  const int ic = list[fic];  // Get filtered cell ID

  // Load up the cell nodes into shared memory

  __shared__ double xv[nDims*nvert];
  __shared__ unsigned char sflag[1];
  __shared__ float dist[7]; // For final min-reduction across block (max sOrderC = 6)

  if (idx == 0)
    sflag[0] = 0;

  for (int i = idx; i < nvert*nDims; i += blockDim.x)
  {
    int d = i % nDims;
    int v = i / nDims;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  __syncthreads();

  if (idx >= 3*nTriFace) return;

  double bboxC[2*nDims];
  double xcC[3];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv,xcC);

  /* ---- Check against our reduced list of faces ---- */

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 1e-2*href;

  float myDist = BIG_DOUBLE;
  double myNorm[3];
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;

  double fxv[nvertf*nDims];

  for (int F = 0; F < NF2; F++)
  {    
    if (myFlag == DC_CUT) continue;

    int ff = checkFaces[ncells*F+fic];
    if (ff < 0)
      continue;

    // Only checking half the element's faces; figure out which ones
    const char corner = corners[ncells*F+fic];
    const char fList[3] = {(char)(corner / 4), (char)(((corner + 1)%4) / 2 + 2), (char)(((corner%4) / 2) + 4)};

    // Get the specific sub-quadrilateral-triangle we're checking here
    const unsigned char fid = fList[idx / (nTriFace)];

    /* ---- Get our specific triangle ---- */

    double TC[9];

    // NOTE: Structured ordering  |  btm,top,left,right,front,back
    const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                         {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

    int I, J, K;
    switch (fid)
    {
      case 0: // Bottom
        I = qid / sOrderC;
        J = qid % sOrderC;
        K = 0;
        break;
      case 1: // Top
        I = qid / sOrderC;
        J = qid % sOrderC;
        K = sOrderC - 1;
        break;
      case 2: // Left
        I = 0;
        J = qid / sOrderC;
        K = qid % sOrderC;
        break;
      case 3: // Right
        I = sOrderC - 1;
        J = qid / sOrderC;
        K = qid % sOrderC;
        break;
      case 4: // Front
        I = qid / sOrderC;
        J = 0;
        K = qid % sOrderC;
        break;
      case 5: // Back
        I = qid / sOrderC;
        J = sOrderC - 1;
        K = qid % sOrderC;
        break;
    }

    int i0 = I+nSideC*(J+nSideC*K);
    int j0 = i0 + nSideC*nSideC;
    int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
    for (int i = 0; i < 8; i++)
      lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

    for (int p = 0; p < 3; p++)
    {
      int ipt = lin2curv[TriPts[fid+tid][p]];
      for (int d = 0; d < 3; d++)
        TC[3*p+d] = xv[3*ipt+d];
    }

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    // 1) In case of face entirely inside element, check if a pt is inside ele
    if (idx == 0)
    {
      double bboxF[6];
      cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

      if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
      {
        double rst[3];
        if (mb.getRefLoc<nSideC>(xv, bboxC, fxv, rst))
          sflag[0] = DC_CUT;
      }
    }

    __syncthreads();

    if (sflag[0] == DC_CUT)
    {
      if (idx == 0)
        cutFlag[ic] = (cutType != 0) ? DC_HOLE : DC_NORMAL;

      return;
    }

    // Find distance from face to cell
    dPoint vec;
    double dist = intersectionCheckOne<nSideC,nSideF>(mb, fxv, &vec[0], TC);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (dist < dtol) // They intersect 1e-8*href
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .02*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .02*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from our element; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }
  }

  if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  /* ---- Synchronize within element ---- */

  float minDist = warpAllReduceMin(myDist);

  int lane = idx % warpSize;
  int wid = idx / warpSize;

  if (lane == 0) dist[wid] = minDist;

  __syncthreads();

  minDist = (idx < blockDim.x / warpSize) ? dist[lane] : 0;

  if (wid == 0)
  {
    minDist = warpAllReduceMin(minDist);
    dist[0] = minDist;
  }

  __syncthreads();

  minDist = dist[0];

  // Thread with minimum ('best') distance sets final cutting flag
  // Race condition if multiple threads have dist 0, but then all will be
  // setting the same value ('DC_CUT') anyways
  if (myDist == minDist)
    cutFlag[ic] = myFlag;
}

__global__
void getElementBoundingBoxes(dMeshBlock mb, dvec<int> eleList, int nEles,
    dvec<float> eleBbox)
{
  int IC = blockIdx.x * blockDim.x + threadIdx.x;

  if (IC >= nEles) return;

  int ic = eleList[IC];

  float xv[8*3];  // Only concerning ourselves with linear portion of ele
  for (int i = 0; i < 8; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = mb.coord[ic+mb.ncells*(d+3*i)];

  cuda_funcs::getBoundingBox<3,8>(xv,&eleBbox[6*IC]);
}

template<int nSideC, int nSideF>
__global__
void cuttingPass3One(dMeshBlock mb, dvec<float> cutFaces, dvec<int> checkFaces,
    dvec<int> list, int nEles, dvec<float> eleBbox,
    dvec<char> corners, dvec<float> outDist, dvec<float> outVec)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int FID = threadIdx.y;

  const int nDims = 3;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;
  const int sOrderC = nSideC - 1;

  const int nQuadFace = sOrderC*sOrderC;
  const int nTriFace = 2*nQuadFace;
  const int nTri = 3*nTriFace;

  /* --- Get our specific element & sub-triangle of element --- */

  const int IC = tid / (3 * nTriFace);
  const int T = tid % (3 * nTriFace);

  const unsigned char q = (T % (nTriFace)) / 2;
  const unsigned char t = (T % (nTriFace)) % 2;
  char F = T / nTriFace;

  if (IC >= nEles) return;

  const int ic = list[IC];  // Get filtered cell ID
  const int ff = checkFaces[nEles*FID+IC];

  if (ff < 0)
    return;

  float bboxC[2*nDims];
  for (int i = 0; i < 2*nDims; i++)
    bboxC[i] = eleBbox[6*ic+i];

  /* ---- Check against our reduced list of faces ---- */

  /// Or TODO on storing href in global memory instead...
  const float href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const float dtol = 2e-2*href;

  // Only checking half the element's faces; figure out which ones
  const char corner = corners[nEles*FID+IC];

  switch (F)
  {
    case 0:
      F = corner / 4;
      break;
    case 1:
      F = ((corner + 1)%4) / 2 + 2;
      break;
    case 2:
      F = (((corner%4) / 2) + 4);
      break;
  }

  /* ---- Get our specific triangle ---- */

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
    {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  int I, J, K;
  switch (F)
  {
    case 0: // Bottom
      I = q / sOrderC;
      J = q % sOrderC;
      K = 0;
      break;
    case 1: // Top
      I = q / sOrderC;
      J = q % sOrderC;
      K = sOrderC - 1;
      break;
    case 2: // Left
      I = 0;
      J = q / sOrderC;
      K = q % sOrderC;
      break;
    case 3: // Right
      I = sOrderC - 1;
      J = q / sOrderC;
      K = q % sOrderC;
      break;
    case 4: // Front
      I = q / sOrderC;
      J = 0;
      K = q % sOrderC;
      break;
    case 5: // Back
      I = q / sOrderC;
      J = sOrderC - 1;
      K = q % sOrderC;
      break;
  }

  int i0 = I+nSideC*(J+nSideC*K);
  int j0 = i0 + nSideC*nSideC;
  int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
  for (int i = 0; i < 8; i++)
    lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

  float TC[9];
  for (int p = 0; p < 3; p++)
  {
    int ipt = lin2curv[TriPts[F+t][p]];
    for (int d = 0; d < 3; d++)
      TC[3*p+d] = mb.coord[ic+mb.ncells*(d+nDims*ipt)]; /// NOTE: 'row-major' ZEFR layout
  }

  // Find distance from face to cell
  /// NOTE: ignoring case of face entirely inside cell, since any valid grid
  /// will also have a different face which intersects its boundary
  dPointf vec;
  double myDist = intersectionCheckOne<nSideC,nSideF>(mb, &cutFaces[ff*stride], &vec[0], TC);
  vec /= vec.norm();

  if (myDist < dtol) // They intersect
  {
    myDist = 0.;
  }

  // Write out results to global memory for future reduction
  outDist[T+nTri*(IC+nEles*FID)] = myDist;
  for (int i = 0; i < 3; i++)
  {
    //outNorm[IC+nEles*(i+3*FID)] = myNorm[i];
    outVec[T+nTri*(IC+nEles*(FID+NF2*i))] = vec[i];
  }
}

__global__
void getMinDist(dvec<float> dists, dvec<float> vecs, int nEles, int nTri)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;
  const int F = threadIdx.y;

  if (IC >= nEles) return;

  // Find minimum tri-face distance for this face/element
  float minDist = BIG_FLOAT;
  float myVec[3] = {0.0f};
  for (int i = 0; i < nTri; i++)
  {
    float dist = dists[i+nTri*(IC+nEles*F)];

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        myVec[d] = vecs[i+nTri*(IC+nEles*(F+NF2*d))];
    }
  }

  // NOTE: Assuming NF2 always <= nTri
  dists[F+nTri*IC] = minDist;
  for (int d = 0; d < 3; d++)
    vecs[F+nTri*(IC+nEles*NF2*d)] = myVec[d];
}

__global__
void getFinalFlag(dvec<int> eleList, dvec<int> checkFaces,
    dvec<float> cutFaces, int nEles, int nvertf, int nTri, dvec<int> cutFlag,
    dvec<float> dists, dvec<float> vecs, int cutType)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;

  if (IC >= nEles) return;

  const int ic = eleList[IC];

  const float dtol = 1e-5; /// TODO: load xv or bboxC & calculate href...?

  // Find nearest face distance for this element

  int nMin = 0;
  char myFlag = DC_UNASSIGNED;
  float minDist = BIG_FLOAT;
  dPointf myNorm;

  for (int F = 0; F < NF2; F++)
  {
    int ff = checkFaces[nEles*F+IC];
    dPointf norm = faceNormal(&cutFaces[ff*nvertf*3]);

    float dist = dists[F+nTri*IC];

    dPointf vec;
    for (int d = 0; d < 3; d++)
      vec[d] = vecs[F+nTri*(IC+nEles*NF2*d)];

    if (dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      minDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (minDist - .02*dtol))
    {
      // Unflagged cell, or have a closer face to use
      minDist = dist;
      myNorm = norm;
      double dot = myNorm*vec;

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - minDist) <= .02*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      minDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
  }

  myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  // Write out final result
  cutFlag[ic] = myFlag;
}

/// TODO: write kernel to do final sort & flag assignment for each element

/*! Remove all elements which do not intersect with cut group's bbox from
 *  consideration (obviously do not intersect) */
template<int nvert>
__global__
void filterElements(dMeshBlock mb, dvec<double> cut_bbox, dvec<int> filt,
    dvec<int> cutFlag, dvec<int> nfilt, dvec<float> bboxOut, int cutType)
{
  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ic == 0)
  {
    nfilt[0] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1.e10f;
      bboxOut[d+3] = -1.e10f;
    }
  }

  __shared__ float bboxF[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxF[i] = (float)cut_bbox[i];

  __syncthreads();

  if (ic >= mb.ncells) return;

  // Set all cell flags initially to DC_NORMAL (filtered cells will remain 'NORMAL')
  cutFlag[ic] = DC_NORMAL;

  float href = .005/3.*(bboxF[3]-bboxF[0]+bboxF[4]-bboxF[1]+bboxF[5]-bboxF[2]);

  // Get element nodes
  float xv[nvert*3];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = (float)mb.coord[ic+mb.ncells*(d+3*i)];

  // Get element bounding box
  float bboxC[6], xc[3];
  cuda_funcs::getBoundingBox<3,nvert>(xv, bboxC);

  bool checkH = false;

  if (cutType == 1)
  {
    // Wall boundary - set as hole if centroid inside wall
    cuda_funcs::getCentroid<3,nvert>(xv, xc);
    char tag = cuda_funcs::checkHoleMap(xc, mb.hm_sam.data(), mb.hm_nx.data(), mb.hm_extents.data());
    checkH = (tag != 1);
  }
  else
  {
    // Overset boundary - only set as hole if entirely inside hole map
    for (int i = 0; i < 8; i++)
    {
      for (int d = 0; d < 3; d++)
        xc[d] = xv[3*i+d];

      if (mb.rrot) // Transform xc to hole map's coordinate system
      {
        double x2[3] = {0.,0.,0.};
        for (int d1 = 0; d1 < 3; d1++)
          for (int d2 = 0; d2 < 3; d2++)
            x2[d1] += mb.Rmat[d1+3*d2]*(xc[d2]-mb.offset[d2]);

        for (int d = 0; d < 3; d++)
          xc[d] = x2[d];
      }

      char tag = cuda_funcs::checkHoleMap(xc, mb.hm_sam.data(), mb.hm_nx.data(), mb.hm_extents.data());
      checkH = checkH || (tag != 1);
    }
  }

  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, href);

  // If filtering element due to being completely inside hole region, tag as hole
  if (!checkH)
    cutFlag[ic] = DC_HOLE;

  if (checkH && checkB)
  {
    int ind = atomicAggInc(&nfilt[0]);
    filt[ind] = ic;
    for (int d = 0; d < 3; d++)
    {
      atomicMinf(&bboxOut[d], bboxC[d]);
      atomicMaxf(&bboxOut[d+3], bboxC[d+3]);
    }
  }
}

/*! Remove all cutting faces which do not intersect this rank's reduced bbox
 *  from consideration (obviously do not intersect) */
template<int nvertf>
__global__
void filterFaces(dvec<float> ele_bbox, int nCut,
    dvec<float> cutFaces, dvec<int> filt, dvec<int> nfilt, dvec<float> bboxOut)
{
  const unsigned int ff = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ff == 0)
  {
    nfilt[1] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1e10f;
      bboxOut[d+3] = -1e10f;
    }
  }

  __shared__ float bboxE[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxE[i] = ele_bbox[i];

  __syncthreads();

  if (ff >= nCut) return;

  float href = .01f/3.f*(bboxE[3]-bboxE[0]+bboxE[4]-bboxE[1]+bboxE[5]-bboxE[2]);

  // Get face nodes
  float fxv[nvertf*3];
  for (int i = 0; i < nvertf; i++)
    for (int d = 0; d < 3; d++)
      fxv[3*i+d] = (float)cutFaces[(ff*nvertf+i)*3+d];

  // Get face bounding box
  float bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxF, bboxE, href);

  if (checkB)
  {
    int ind = atomicAggInc(&nfilt[1]);
    filt[ind] = ff;
    for (int d = 0; d < 3; d++)
    {
       atomicMinf(&bboxOut[d], bboxF[d]);
       atomicMaxf(&bboxOut[d+3], bboxF[d+3]);
    }
  }
}

void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, double *cutBbox_h, int* cutFlag, int cutType)
{
  // Setup cutMap TODO: create initialization elsewhere?
  cutFlag_d.resize(ncells);
  filt_eles.resize(ncells);
  filt_faces.resize(nCut);

//  dvec<double> cutFaces;
//  cutFaces.assign(cutFaces_h, nCut*nvertf*nDims);
  std::vector<float> cutFaces_hf(nCut*nvertf*nDims);
  for (int i = 0; i < nvertf*nDims*nCut; i++)
    cutFaces_hf[i] = (float)cutFaces_h[i];

  dvec<float> cutFaces;
  cutFaces.assign(cutFaces_hf.data(), nCut*nvertf*nDims);

  dvec<double> cutBbox_d;
  cutBbox_d.assign(cutBbox_h, 2*nDims);

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);
  }

  /* Filter elements based upon cutting-surface bounding box & Cartesian approx. rep. */

  hvec<int> nfilt_h;
  dvec<int> nfilt_d;
  nfilt_h.resize(2);
  nfilt_h[0] = 0;  nfilt_h[1] = 0;
  nfilt_d.assign(nfilt_h.data(), nfilt_h.size());

  ele_bbox.resize(6);
  face_bbox.resize(6);

  int threads = 128;
  int blocks = (ncells + threads - 1) / threads;

  switch(nvert)
  {
    case 8:
      filterElements<8><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    case 27:
      filterElements<27><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    case 64:
      filterElements<64><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    default:
      printf("nvert = %d\n",nvert);
      ThrowException("nvert case not implemented for filterElements on device");
  }

  check_error();

  if (nCut == 0)
  {
    cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

    cutFaces.free_data();
    cutBbox_d.free_data();
    nfilt_h.free_data();
    nfilt_d.free_data();

    return;
  }

  /* Filter cutting faces by intersection with the filtered elements' bounding box */

  blocks = (nCut + threads - 1) / threads;

  switch(nvertf)
  {
    case 4:
      filterFaces<4><<<blocks, threads, 6*sizeof(float)>>>(ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    case 16:
      filterFaces<16><<<blocks, threads, 6*sizeof(float)>>>(ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    default:
      printf("nvertf = %d\n",nvertf);
      ThrowException("nvertf case not implemented for filterFaces on device");
  }

  check_error();

  nfilt_h.assign(nfilt_d.data(), 2);
  int nfiltC = nfilt_h[0];
  int nfiltF = nfilt_h[1];
  printf("%d: nfilt = %d/%d, %d/%d; cutType = %d\n",rank,nfilt_h[0], ncells, nfilt_h[1], nCut, cutType);

  /* Perform the Direct Cut algorithm on the filtered list of elements & faces */

  dvec<int> checkFaces;
  dvec<char> corners;
  checkFaces.resize(NF2*nfiltC);
  corners.resize(NF2*nfiltC);

  int blocks1 = nfiltC;
  int threads1 = 32;
  int nbShare1 = (8*nDims)*sizeof(double) + 32*sizeof(float) + (32+1)*sizeof(int) + 32*sizeof(char);

  int nSideC = std::cbrt(nvert);
  int nTri = 3*2*(nSideC-1)*(nSideC-1);
  int nbShare3 = nDims*nvert*sizeof(double) + 7*sizeof(float) + 1;
  
  /// TODO: new approach...
  dvec<float> cfDist;
  dvec<char> cfCorner;
  cfDist.resize(nfiltC*nfiltF);
  cfCorner.resize(nfiltC*nfiltF);

  // Sort the distance lists for each element [down to NF2 faces to check in detail]

//  dim3 threads0(32,4);
//  dim3 blocks0( (nfiltC + threads0.x - 1) / threads0.x, (nfiltF + threads0.y - 1) / threads0.y );

//  dim3 threads3(32,NF2);
//  blocks = (nfiltC*3*nTri + threads3.x - 1) / threads3.x;
//  cuttingPass3One<<<blocks0,threads3>>>(*this, cutFaces, nvertf, nfiltF, filt_faces,
//      nfiltF, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces,
//      corners, cfDist, cfNorm);

  /* Pass 1: 'Coarse-Grained' Check using Bounding Boxes
   * Pass 2: 'Medium-Grained' Check using Linear Parts of Elements/Faces
   * Pass 3: 'Finest-Grained' Direct Cut Check
   */

  if (nfiltC > 0)
  {
//    cuttingPass1<<<blocks1, threads1, nbShare1>>>(*this, cutFaces, nvertf, nfiltF,
//        filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfiltC, checkFaces, corners);
//    check_error();


    dim3 Threads1(32,4);
    dim3 Blocks1( (nfiltC + Threads1.x - 1) / Threads1.x,
                  (nfiltF + Threads1.y - 1) / Threads1.y );

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(getElementBoundingBoxes), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass1New), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(sortFaces), hipFuncCachePreferL1);

    dvec<float> eleBbox;
    eleBbox.resize(6*nfiltC);

    int ThreadsB = 128;
    int BlocksB = (nfiltC + ThreadsB - 1) / ThreadsB;

    getElementBoundingBoxes<<<BlocksB,ThreadsB>>>(*this, filt_eles, nfiltC, eleBbox);

    // Have each filtered element calculate a rough distance to each filtered face
    cuttingPass1New<<<Blocks1,Threads1>>>(*this, filt_eles, nfiltC, cutFaces,
        nvertf, nfiltF, filt_faces, cfCorner, cfDist);
    check_error();

    int ThreadsS = 128;
    int BlocksS = (nfiltC + ThreadsS - 1) / ThreadsS;
    sortFaces<<<BlocksS, ThreadsS>>>(filt_faces, nfiltC, nfiltF, cfDist,
        cfCorner, checkFaces, corners);
    check_error();

    dvec<float> cfNorm, cfVec;
    cfNorm.resize(3*nfiltC*nTri*NF2);
    cfVec.resize(3*nfiltC*nTri*NF2);
    cfDist.resize(nfiltC*nTri*NF2);


    dim3 t3(32,4);
    int b3 = (nfiltC*nTri + t3.x - 1) / t3.x;

    switch(nvertf)
    {
      case 4:
        switch(nvert)
        {
          case 8:
            // Both element & face linear - no further checking required
            break;
          default:
            printf("nvert = %d\n",nvert);
            ThrowException("nvert case not implemented for directCut on device");
        }
        break;

      case 16:
        switch(nvert)
        {
          case 8:
//            cuttingPass3<2,4><<<blocks1, nTri, nbShare3>>>(*this, cutFaces, checkFaces, cutFlag_d.data(), cutType, filt_eles, nfiltC, corners);
            check_error();

            cuttingPass3One<2,4><<<b3, t3>>>(*this, cutFaces, checkFaces, filt_eles, nfiltC, eleBbox, corners, cfDist, cfVec);
            check_error();
            break;
          case 64:
//            cuttingPass3<4,4><<<blocks1, nTri, nbShare3>>>(*this, cutFaces, checkFaces, cutFlag_d.data(), cutType, filt_eles, nfiltC, corners);
            check_error();

            cuttingPass3One<4,4><<<b3, t3>>>(*this, cutFaces, checkFaces, filt_eles, nfiltC, eleBbox, corners, cfDist, cfVec);
            check_error();
            break;
          default:
            printf("nvert = %d\n",nvert);
            ThrowException("nvert case not implemented for directCut on device");
        }
        break;
      default:
        printf("nvertFace = %d, nCut = %d\n",nvertf,nCut);
        ThrowException("nvertFace case not implemented for directCut on device");
    }

    int BlocksM = (nfiltC + 32 - 1) / 32;
    getMinDist<<<BlocksM, t3>>>(cfDist, cfVec, nfiltC, nTri);
    getFinalFlag<<<BlocksM, 128>>>(filt_eles, checkFaces, cutFaces, nfiltC, nvertf,
        nTri, cutFlag_d, cfDist, cfVec, cutType);

    // Free data...
    cfNorm.free_data();
    cfVec.free_data();
    eleBbox.free_data();
  }

  cfDist.free_data();
  cfCorner.free_data();

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  // Free all data allocated in this function

  nfilt_d.free_data();
  nfilt_h.free_data();

  corners.free_data();
  checkFaces.free_data();

  cutFaces.free_data();
  cutBbox_d.free_data();
}
