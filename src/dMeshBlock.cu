#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

#include "hip/device_functions.h"
#include "math.h"

/* --- Handy Vector Operation Macros --- */

#define NF1 16 // 20-32 depending on unstructured-ness of grid & desire for robustness
#define NF2  4 // 3-6 depending on unstructured-ness of grid & desire for robustness

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
double DOTCROSS4(const double* __restrict__ c,
                 const double* __restrict__ a1, const double* __restrict__ a2,
                 const double* __restrict__ b1, const double* __restrict__ b2)
{
  double d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}

/* --- Misc. Helpful CUDA kernels --- */

#define WARP_SZ 32

__device__
inline int lane_id(void) { return threadIdx.x % WARP_SZ; }

__device__
inline int warp_bcast(int v, int leader) { return __shfl(v, leader); }

/*! Warp-aggregated atomic increment
 *  https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/ */
__device__
int atomicAggInc(int *ctr)
{
  int mask = __ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  int res;
  if (lane_id() == leader)
    res = atomicAdd(ctr, __popc(mask));
  // brodcast result
  res = warp_bcast(res, leader);
  // each thread computes its own value
  return res + __popc(mask & ((1 << lane_id()) - 1));
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  xlist.assign(xlist_h.data(), xlist_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true;
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
double lineSegmentDistance(double *p1, double *p2, double *p3, double *p4, double *dx)
{
  // Get the line equations
  double U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  double V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  double W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  double uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  double vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  double uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  double uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  double vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  double den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  double s = (den < 1e-10) ? 0 : (uv*vw - vv*uw) / den;
  double t = (den < 1e-10) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.), 1.);
  t = fmin(fmax(t, 0.), 1.);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  double dist = 0;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise 
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
double triTriDistance2(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const double* V01 = T1;
  const double* V11 = T1+3;
  const double* V21 = T1+6;

  const double* V02 = T2;
  const double* V12 = T2+3;
  const double* V22 = T2+6;

  double N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  double norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  double d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  double d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  double d01 = DOT(N2,V01) + d2;
  double d11 = DOT(N2,V11) + d2;
  double d21 = DOT(N2,V21) + d2;

  double d02 = DOT(N1,V02) + d1;
  double d12 = DOT(N1,V12) + d1;
  double d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      double P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      double P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      double P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      double P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      double P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      double P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  double L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  double p0 = DOT(L,V01);
  double p1 = DOT(L,V11);
  double p2 = DOT(L,V21);

  double q0 = DOT(L,V02);
  double q1 = DOT(L,V12);
  double q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

static
__device__ __forceinline__
dPoint faceNormal(const double* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPoint pt0 = dPoint(&xv[0]);
  dPoint pt1 = dPoint(&xv[3]);
  dPoint pt2 = dPoint(&xv[6]);
  dPoint norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPoint(&xv[9]);
  dPoint norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPoint norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

template<int nSideC, int nSideF>
__device__
double intersectionCheck(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec, bool PRINT)
{
  /* --- Prerequisites --- */

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  const int sorderC = nSideC-1;
  const int sorderF = nSideF-1;

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                       {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  double tol = 1e-9;
  double TC[9], TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,nvert>(exv, bboxC);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<nSideC>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Check outer faces of element for intersection with face
#pragma unroll
  for (int f = 0; f < 6; f++)
  {
#pragma unroll
    for (int g = 0; g < sorderC*sorderC; g++)
    {
      int I, J, K;
      switch (f)
      {
        case 0: // Bottom
          I = g / sorderC;
          J = g % sorderC;
          K = 0;
          break;
        case 1: // Top
          I = g / sorderC;
          J = g % sorderC;
          K = sorderC - 1;
          break;
        case 2: // Left
          I = 0;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 3: // Right
          I = sorderC - 1;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 4: // Front
          I = g / sorderC;
          J = 0;
          K = g % sorderC;
          break;
        case 5: // Back
          I = g / sorderC;
          J = sorderC - 1;
          K = g % sorderC;
          break;
      }

      int i0 = I+nSideC*(J+nSideC*K);
      int j0 = i0 + nSideC*nSideC;
      int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
      for (int i = 0; i < 8; i++)
        lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

      // Get triangles for the sub-hex of the larger curved hex
      for (int i = f; i < f+2; i++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = lin2curv[TriPts[i][p]];
          for (int d = 0; d < 3; d++)
            TC[3*p+d] = exv[3*ipt+d];
        }

        cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
        double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
        btol = fmin(btol, minDist);
        if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;

        for (int M = 0; M < sorderF; M++)
        {
          for (int N = 0; N < sorderF; N++)
          {
            int m0 = M + nSideF*N;
            int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
            for (int m = 0; m < 2; m++)
              for (int n = 0; n < 3; n++)
                TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

            // Intersection check between element face tris & cutting-face tris
            for (int j = 0; j < 2; j++)
            {
              for (int p = 0; p < 3; p++)
              {
                int ipt = TriPtsF[j][p];
                for (int d = 0; d < 3; d++)
                  TF[3*p+d] = fxv[3*ipt+d];
              }

              double vec[3];
              double dist = triTriDistance2(TF, TC, vec, tol);

              if (dist < tol)
                return 0.;

              if (dist < minDist)
              {
                for (int d = 0; d < 3; d++)
                  minVec[d] = vec[d];
                minDist = dist;
              }
            }
          }
        }
      }
    }
  }

  // 3) Definitely no intersection; use centroids to get vector
  if (minDist == BIG_DOUBLE)
  {
    double tmp[3];
    cuda_funcs::getCentroid<3,nvert>(exv,minVec);
    cuda_funcs::getCentroid<3,nvertf>(fxv,tmp);

    minDist = 0;
    for (int d = 0; d < 3; d++) // Vector is face -> cell
    {
      minVec[d] -= tmp[d];
      minDist += minVec[d]*minVec[d];
    }

    return sqrt(minDist);
  }

  return minDist;
}

__device__
double intersectionCheckLinear(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec, bool PRINT)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  double tol = 1e-8f;
  double TC[9], TF[9];
  double minDist = 1e15;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,4>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,8>(exv, bboxC);

  double xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<2>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Find nearest corner of element to face; check only that half of element

  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    double dist = 0.;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        minVec[d] = exv[3*i+d] - xcf[d];
      corner = i;
    }
  }

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const int fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

  // 3) Check those faces of element for intersection with face
#pragma unroll
  for (int F = 0; F < 3; F++)
  { 
    int f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        int ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      /*cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
      double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
      btol = fmin(btol, minDist);
      if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;*/

      int TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

template<int nDims, int nSideC, int nSideF>
__global__
void fillCutMap(dMeshBlock mb, dvec<double> cutFaces, int nCut,
    int* __restrict__ cutFlag, int cutType, int* __restrict__ list, int ncells)
{
  int ic = blockIdx.x * blockDim.x + threadIdx.x;

  if (ic >= ncells) return;

  ic = list[ic];  // Get filtered cell ID

  // Figure out how many threads are left in this block after ic>=ncells returns
  int blockSize = min(blockDim.x, ncells - blockIdx.x * blockDim.x);

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;

  const bool LINEAR = (nSideC == 2 && nSideF == 2);

  //bool PRINT = threadIdx.x == 0 && blockIdx.x == 0;

  // Load up the cell nodes into an array
  double xv[nDims*nvert];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < nDims; d++)
      xv[nDims*i+d] = mb.coord[ic+mb.ncells*(d+nDims*i)]; /// NOTE: 'row-major' ZEFR layout

  double bboxC[2*nDims];
  double xcC[3];
  double xcFf[3];
  float oobb[15];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv,xcC);

  bool PRINT = false; //(fabs(xcC[0]-.71) < .08) && (fabs(xcC[1]+.56) < .08) && (fabs(xcC[2]+.56) < .08);

  cuda_funcs::getOOBB<nDims,nvert>(xv,oobb,PRINT);

  /// DEBUGGING
  if (PRINT)
  {
    printf("%d: Nodes 0/6 = %f, %f, %f;  %f, %f, %f\n",ic,xv[0],xv[1],xv[2],xv[18],xv[19],xv[20]);
    printf("%d: OOBB axes = %f, %f, %f;  %f, %f, %f;  %f, %f, %f\n",ic,oobb[0],oobb[1],oobb[2],oobb[3],oobb[4],oobb[5],oobb[6],oobb[7],oobb[8]);
    printf("%d: OOBB min/max pt = %f, %f, %f;  %f, %f, %f\n",ic,oobb[9],oobb[10],oobb[11],oobb[12],oobb[13],oobb[14]);
  }

  /*float xcRc[3];
  for (int d = 0; d < 3; d++)
    xcRc[d] = 0.5f * (oobb[9+d] + oobb[12+d]);*/

  /// DEBUGGING
  /*double RAD1 = .061;
  bool PRINT = (ic == 23776) && (nSideC==2) && (fabs(xcC[0]+.0131) < .02*RAD1) && (fabs(xcC[1]-.0271) < .02*RAD1) && (fabs(xcC[2]-.0016) < .015*RAD1);
  PRINT = false;
  if (PRINT)
  {
    printf("Found cell = %d/%d, filtered ID %d, XC %.4e %.4e %.4e\n",ic,ncells,blockIdx.x * blockDim.x + threadIdx.x,xcC[0],xcC[1],xcC[2]);

    for (int i =0; i<8; i++)
    {
      printf("node %d = %f, %f, %f\n",i,xv[i*3+0],xv[i*3+1],xv[i*3+2]);
    }
  }*/

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 2e-1*href;

  /* --- Pass 1: Coarse-Grained Distance Calculation Using Bounding Boxes --- */

  double distList[NF1];
  int faceList[NF1];
  short nHit = 0;
  __shared__ double fxv_s[nDims*4];
  float fxv_r[nDims*4] = {0.0f};

  for (int i = 0; i < NF1; i++)
  {
    distList[i] = BIG_DOUBLE;
    faceList[i] = -1;
  }

  for (int ff = 0; ff < nCut; ff++)
  {
    __syncthreads();

    // Stick to just linear component of face [4 corners]
    for (int i = threadIdx.x; i < 4*nDims; i += blockSize)
      fxv_s[i] = cutFaces[ff*stride+i];

    __syncthreads();

    // Transform face points to element OOBB axes
    for (int i = 0; i < 12; i++)
      fxv_r[i] = 0.f;

    for (int k = 0; k < 4; k++)
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
          fxv_r[3*k+i] += oobb[3*i+j] * fxv_s[3*k+j];

    // Get OBB of face in element's axes
    float obbF[6];
    cuda_funcs::getBoundingBox<nDims,4>(fxv_r,obbF);

    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcFf);

    /*
    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcF);

    double dist2 = 0.;
    for (int d = 0; d < 3; d++)
      dist2 += (xcC[d] - xcF[d]) * (xcC[d] - xcF[d]);*/

    float dist1 = cuda_funcs::boundingBoxDist<3>(&oobb[9], &obbF[0]);
    //bool check = cuda_funcs::boundingBoxCheck<3>(oobb,obbF,.01f*(float)dtol);
    bool check = (dist1 <= .01f*(float)dtol);

    double dist2 = 0;
    for (int d = 0; d < 3; d++)
      dist2 += (xcFf[d] - xcC[d]) * (xcFf[d] - xcC[d]);
    dist2 = 0.5*(sqrt(dist2) + dist1);

    if (check)
    {
      // Build up list of faces with bbox intersect from the front
      nHit++;

      int ind = min(NF1 - 1, nHit);

      if (dist2 > distList[ind]) continue;

      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > 0)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }
    else
    {
      int ind = NF1 - 1;
      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > nHit)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }

    /*if (nHit == 0)
    {
      if (check)
      {
        distList[0] = dist2;
        faceList[1] = ff;
        nHit = 1;
      }
      if (dist2 < distList[NF1-1]) // Closer than the worst in our list, at least
      {
        // Insert and sort
        distList[NF1-1] = dist2;
        faceList[NF1-1] = ff;
        int ind = NF1 - 1;
        while (distList[ind] < distList[ind-1] && ind > 0)
        {
          swap(distList[ind], distList[ind-1]);
          swap(faceList[ind], faceList[ind-1]);
          ind--;
        }
      }
    }
    else if (check)
    {
      nHit++;

      int ind = min(NF1 - 1, nHit);
      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > 0)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }

      /*int ind = 0;
      for (ind = 0; ind < nHit; ind++)
      {
        if (dist2 < distList[ind])
        {
          for (int j = nHit; j > ind; j--)
          {
            distList[j] = distList[j-1];
            faceList[j] = faceList[j-1];
          }
          break;
        }
      }
      distList[ind] = dist2;
      faceList[ind] = ff;
      nHit++;* /
    }*/

    if (PRINT && dist2 < 0.3f)
    {
      for (int i = 0; i < 4; i++)
        printf("Cell %d: Face[%d][%d] = %f %f %f\n",ic,ff,i,fxv_s[3*i+0],fxv_s[3*i+1],fxv_s[3*i+2]);
      for (int i = 0; i < 4; i++)
        printf("Cell %d: FaceR[%d][%d] = %f %f %f\n",ic,ff,i,fxv_r[3*i+0],fxv_r[3*i+1],fxv_r[3*i+2]);
      printf("Cell %d face %d distance estimate %.4e\n",ic,ff,dist2);
    }

    /*if (dist2 < distList[NF-1]) // Closer than the worst in our list, at least
    {
      // Insert and sort
      distList[NF-1] = dist2;
      faceList[NF-1] = ff;
      int ind = NF-1;
      while (distList[ind] < distList[ind-1] && ind > 0)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }*/
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

  int ncheck = (nHit > 0) ? min(nHit,NF1) : NF2;

  int faces[NF2];
  for (int i = 0; i < NF2; i++)
  {
    faces[i] = -1;
    distList[i] = 1e15;
  }

  double fxv[nDims*nvertf];
  double myNorm[3] = {0., 0., 0.};
  double myDist = BIG_DOUBLE;
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;

  for (int F = 0; F < ncheck; F++)
  {
    if (LINEAR && myFlag == DC_CUT) continue;

    int ff = faceList[F];
    if (ff < 0)
      continue;

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    dPoint vec;
    double dist = intersectionCheck<2,2>(mb, fxv, xv, &vec[0], PRINT);
    //double dist = intersectionCheckLinear(mb, fxv, xv, &vec[0], PRINT);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (LINEAR && dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .1*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .1*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from surface; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }

    if (!LINEAR)
    {
      // Insert face into final sorted list of faces to check more fully
      if (dist < distList[NF2-1])
      {
        // Insert and sort
        distList[NF2-1] = dist;
        faces[NF2-1] = ff;
        int ind = NF2 - 1;
        while (distList[ind] < distList[ind-1] && ind > 0)
        {
          swap(distList[ind], distList[ind-1]);
          swap(faces[ind], faces[ind-1]);
          ind--;
        }
      }
    }
  }

  /*if (nHit == 0) /// dunno... use method above to avoid too much divergence...
  {
    // Definitely no intersection; just stick with top NF2 faces
    for (int i = 0; i < NF2; i++)
      faces[i] = faceList[i];
  }
  else
  {

  }*/

  /* --- 6/24/17
     + Do 'intersectionCheck()' on linear component of cell/face only
     ++ Do for all ncheck faces
     + If nSideC and nSideF == 2, return this result
     + If nSideC or nSideF > 2, do 'full' intersectionCheck() using
       the nearest half of the element
     --- */

  // If entirely linear grid system, call it good & return
  if (LINEAR)
  {
    if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

    cutFlag[ic] = myFlag;
    return;
  }

  /* --- Pass 3: More-Accurate Distace Calculate Using Triangles --- */

  // Reset variables
  myDist = BIG_DOUBLE;
  myFlag = DC_UNASSIGNED;
  nMin = 0;

  for (int F = 0; F < NF2; F++)
  {
    if (myFlag == DC_CUT) continue;

    int ff = faces[F];
    if (ff < 0)
      continue;

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    /// DEBUGGING
    if (PRINT)
    {
      for (int i = 0; i < nvertf; i++)
        printf("Cell %d: Face[%d][%d] = %f %f %f\n",ic,ff,i,fxv[3*i+0],fxv[3*i+1],fxv[3*i+2]);
    }

    // Find distance from face to cell
    dPoint vec;
    double dist = intersectionCheck<nSideC,nSideF>(mb, fxv, xv, &vec[0], PRINT);
    vec /= vec.norm();

    if (PRINT)
    {
      printf("Face %d: dist = %.4e, vec = %f, %f, %f\n",ff,dist,vec[0],vec[1],vec[2]);
    }

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (dist < dtol) // They intersect 1e-8*href
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .02*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .02*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from surface; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }
    if (PRINT) printf("After face %d-%d: Cut Flag = %d\n",F,ff,myFlag);
  }

  if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  cutFlag[ic] = myFlag;
}

/*! Remove all elements which do not intersect with cut group's bbox from
 *  consideration (obviously do not intersect) */
template<int nvert>
__global__
void filterElements(dMeshBlock mb, dvec<double> cut_bbox, dvec<int> filt, dvec<int> cutFlag, dvec<int> nfilt)
{
  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;

  if (ic >= mb.ncells) return;

  nfilt[0] = 0; // Race condition, but all same value... whatever...
  // Set all cell flags initially to DC_NORMAL (filtered cells will remain 'NORMAL')
  cutFlag[ic] = DC_NORMAL;

  // Figure out how many threads are left in this block after ic>ncells returns
  int blockSize = min(blockDim.x, mb.ncells - blockIdx.x * blockDim.x);

  __shared__ double bboxF[6];
  for (int i = threadIdx.x; i < 6; i += blockSize)
    bboxF[i] = cut_bbox[i];

  __syncthreads();

  double href = .005/3.*(bboxF[3]-bboxF[0]+bboxF[4]-bboxF[1]+bboxF[5]-bboxF[2]);

  // Get element nodes
  double xv[nvert*3];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = mb.coord[ic+mb.ncells*(d+3*i)];

  // Get element bounding box
  double bboxC[3];
  cuda_funcs::getBoundingBox<3,nvert>(xv, bboxC);

  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, href))
    filt[atomicAggInc(&nfilt[0])] = ic;
}

void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, double *cutBbox_h, int* cutFlag, int cutType)
{
  // Setup cutMap TODO: create initialization elsewhere?
  cutFlag_d.resize(ncells);
  filt_list.resize(ncells);

  dvec<double> cutFaces;
  cutFaces.assign(cutFaces_h, nCut*nvertf*nDims);

  dvec<double> cutBbox_d;
  cutBbox_d.assign(cutBbox_h, 2*nDims);

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);
  }

  // Filter elements based upon cutting-surface bounding box

  hvec<int> nfilt_h;
  dvec<int> nfilt_d;
  nfilt_h.resize(1);  nfilt_h[0] = 0;
  nfilt_d.assign(nfilt_h.data(), 1);

  int threads = 128;
  int blocks = (ncells + threads - 1) / threads;

  switch(nvert)
  {
    case 8:
      filterElements<8><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
      break;
//    case 27:
//      filterElements<27><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
//    case 64:
//      filterElements<64><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
//    case 125:
//      filterElements<125><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
    default:
      printf("nvert = %d\n",nvert);
      ThrowException("nvert case not implemented for directCut on device");
  }
hipDeviceSynchronize();
check_error();
  nfilt_h.assign(nfilt_d.data(), 1);

  int nfilt = nfilt_h[0];

  // Perform the Direct Cut algorithm on the filtered list of grid elements
hipDeviceSynchronize();
check_error();
  threads = 32;
  blocks = (nfilt + threads - 1) / threads;
  int nbShare = sizeof(double)*4*nDims;

  if (nfilt > 0)
  {
    switch(nvertf)
    {
//      case 4:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 27:
//            fillCutMap<3,3,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
//      case 9:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 27:
//            fillCutMap<3,3,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
      case 16:
        switch(nvert)
        {
          case 8:
            fillCutMap<3,2,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
            break;
//          case 27:
//            fillCutMap<3,3,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
          default:
            printf("nvert = %d\n",nvert);
            ThrowException("nvert case not implemented for directCut on device");
        }
        break;
//      case 25:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 27:
//            fillCutMap<3,3,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
      default:
        printf("nvertFace = %d\n",nvertf);
        ThrowException("nvertFace case not implemented for directCut on device");
    }
  }
hipDeviceSynchronize();
  check_error();

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  nfilt_d.free_data();
  nfilt_h.free_data();

  cutFaces.free_data();
  cutBbox_d.free_data();
}
