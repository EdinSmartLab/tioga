#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

#include "hip/device_functions.h"
#include "math.h"

/* --- Handy Vector Operation Macros --- */

#define NF1  8 // 20-32 depending on unstructured-ness of grid & desire for robustness
#define NF2  3 // 3-6 depending on unstructured-ness of grid & desire for robustness

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
double DOTCROSS4(const double* __restrict__ c,
                 const double* __restrict__ a1, const double* __restrict__ a2,
                 const double* __restrict__ b1, const double* __restrict__ b2)
{
  double d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}

/* --- Misc. Helpful CUDA kernels --- */

#define WARP_SZ 32

__device__
inline int lane_id(void) { return threadIdx.x % WARP_SZ; }

__device__
inline int warp_bcast(int v, int leader) { return __shfl(v, leader); }

/*! Warp-aggregated atomic increment
 *  https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/ */
__device__
int atomicAggInc(int *ctr)
{
  int mask = __ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  int res;
  if (lane_id() == leader)
    res = atomicAdd(ctr, __popc(mask));
  // brodcast result
  res = warp_bcast(res, leader);
  // each thread computes its own value
  return res + __popc(mask & ((1 << lane_id()) - 1));
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);

  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);
  std::vector<double> xlist_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  xlist.assign(xlist_h.data(), xlist_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::assignHoleMap(bool hasWall, int* nx, int* sam, double* extents)
{
  if (hasWall)
  {
    int size = nx[0]*nx[1]*nx[2];

    std::vector<char> tmp_sam(size);
    for (int i = 0; i < size; i++)
      tmp_sam[i] = (char)sam[i];

    double dx[3];
    for (int d = 0; d < 3; d++)
      dx[d] = (extents[d+3] - extents[d]) / nx[d];

    hm_sam.assign(tmp_sam.data(), size);
    hm_extents.assign(extents, 6);
    hm_nx.assign(nx, 3);
    hm_dx.assign(dx, 3);
  }
  else
  {
    clearHoleMap();
  }
}

void dMeshBlock::clearHoleMap(void)
{
  int nx[3] = {0,0,0};
  double dx[3] = {0,0,0};
  double extents[6] = {0,0,0,0,0,0};

  hm_sam.resize(0);

  hm_nx.assign(nx, 3);
  hm_dx.assign(dx, 3);
  hm_extents.assign(extents, 6);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true;
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
double lineSegmentDistance(double *p1, double *p2, double *p3, double *p4, double *dx)
{
  // Get the line equations
  double U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  double V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  double W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  double uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  double vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  double uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  double uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  double vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  double den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  double s = (den < 1e-10) ? 0 : (uv*vw - vv*uw) / den;
  double t = (den < 1e-10) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fmin(fmax(s, 0.), 1.);
  t = fmin(fmax(t, 0.), 1.);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  double dist = 0;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise 
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
double triTriDistance2(double* T1, double* T2, double* minVec, double tol)
{
  double dist = 1e15;
  double vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      double D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const double* V01 = T1;
  const double* V11 = T1+3;
  const double* V21 = T1+6;

  const double* V02 = T2;
  const double* V12 = T2+3;
  const double* V22 = T2+6;

  double N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  double norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  double d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  double d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  double d01 = DOT(N2,V01) + d2;
  double d11 = DOT(N2,V11) + d2;
  double d21 = DOT(N2,V21) + d2;

  double d02 = DOT(N1,V02) + d1;
  double d12 = DOT(N1,V12) + d1;
  double d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      double P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      double P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      double P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      double P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      double P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      double P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  double L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  double p0 = DOT(L,V01);
  double p1 = DOT(L,V11);
  double p2 = DOT(L,V21);

  double q0 = DOT(L,V02);
  double q1 = DOT(L,V12);
  double q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  double s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  double t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    double dt = fmin(fabs(t1-s2), fabs(s1-t2));
    double dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.;
}

static
__device__ __forceinline__
dPoint faceNormal(const double* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPoint pt0 = dPoint(&xv[0]);
  dPoint pt1 = dPoint(&xv[3]);
  dPoint pt2 = dPoint(&xv[6]);
  dPoint norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPoint(&xv[9]);
  dPoint norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPoint norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

template<int nSideC, int nSideF>
__device__
double intersectionCheck(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec)
{
  /* --- Prerequisites --- */

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;

  const int sorderC = nSideC-1;
  const int sorderF = nSideF-1;

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
                       {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  double tol = 1e-9;
  double TC[9], TF[9];
  double minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,nvert>(exv, bboxC);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<nSideC>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Check outer faces of element for intersection with face
#pragma unroll
  for (int f = 0; f < 6; f++)
  {
#pragma unroll
    for (int g = 0; g < sorderC*sorderC; g++)
    {
      int I, J, K;
      switch (f)
      {
        case 0: // Bottom
          I = g / sorderC;
          J = g % sorderC;
          K = 0;
          break;
        case 1: // Top
          I = g / sorderC;
          J = g % sorderC;
          K = sorderC - 1;
          break;
        case 2: // Left
          I = 0;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 3: // Right
          I = sorderC - 1;
          J = g / sorderC;
          K = g % sorderC;
          break;
        case 4: // Front
          I = g / sorderC;
          J = 0;
          K = g % sorderC;
          break;
        case 5: // Back
          I = g / sorderC;
          J = sorderC - 1;
          K = g % sorderC;
          break;
      }

      int i0 = I+nSideC*(J+nSideC*K);
      int j0 = i0 + nSideC*nSideC;
      int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
      for (int i = 0; i < 8; i++)
        lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

      // Get triangles for the sub-hex of the larger curved hex
      for (int i = f; i < f+2; i++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = lin2curv[TriPts[i][p]];
          for (int d = 0; d < 3; d++)
            TC[3*p+d] = exv[3*ipt+d];
        }

        cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
        double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
        btol = fmin(btol, minDist);
        if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;

        for (int M = 0; M < sorderF; M++)
        {
          for (int N = 0; N < sorderF; N++)
          {
            int m0 = M + nSideF*N;
            int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
            for (int m = 0; m < 2; m++)
              for (int n = 0; n < 3; n++)
                TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

            // Intersection check between element face tris & cutting-face tris
            for (int j = 0; j < 2; j++)
            {
              for (int p = 0; p < 3; p++)
              {
                int ipt = TriPtsF[j][p];
                for (int d = 0; d < 3; d++)
                  TF[3*p+d] = fxv[3*ipt+d];
              }

              double vec[3];
              double dist = triTriDistance2(TF, TC, vec, tol);

              if (dist < tol)
                return 0.;

              if (dist < minDist)
              {
                for (int d = 0; d < 3; d++)
                  minVec[d] = vec[d];
                minDist = dist;
              }
            }
          }
        }
      }
    }
  }

  // 3) Definitely no intersection; use centroids to get vector
  if (minDist == BIG_DOUBLE)
  {
    double tmp[3];
    cuda_funcs::getCentroid<3,nvert>(exv,minVec);
    cuda_funcs::getCentroid<3,nvertf>(fxv,tmp);

    minDist = 0;
    for (int d = 0; d < 3; d++) // Vector is face -> cell
    {
      minVec[d] -= tmp[d];
      minDist += minVec[d]*minVec[d];
    }

    return sqrt(minDist);
  }

  return minDist;
}

__device__
double intersectionCheckLinear(dMeshBlock &mb, const double* __restrict__ fxv,
    const double* __restrict__ exv, double* __restrict__ minVec, bool PRINT)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const unsigned char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  double tol = 1e-8f;
  double TC[9], TF[9];
  double minDist = 1e15;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  double bboxC[6], bboxF[6];
  cuda_funcs::getBoundingBox<3,4>(fxv, bboxF);
  cuda_funcs::getBoundingBox<3,8>(exv, bboxC);

  double xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  // 1) In case of face entirely inside element, check if a pt is inside ele
  if (cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, 0))
  {
    double rst[3];
    if (mb.getRefLoc<2>(exv, bboxC, fxv, rst))
      return 0.;
  }

  // 2) Find nearest corner of element to face; check only that half of element

  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    double dist = 0.;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        minVec[d] = exv[3*i+d] - xcf[d];
      corner = i;
    }
  }

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const int fList[3] = {corner / 4, ((corner + 1)%4) / 2 + 2, ((corner%4) / 2) + 4};

  // 3) Check those faces of element for intersection with face
#pragma unroll
  for (int F = 0; F < 3; F++)
  { 
    int f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = f; i < f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        int ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      /*cuda_funcs::getBoundingBox<3,3>(TC, bboxC);
      double btol = .1*(bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]);
      btol = fmin(btol, minDist);
      if (!cuda_funcs::boundingBoxCheck<3>(bboxC,bboxF,btol)) continue;*/

      int TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        double vec[3];
        double dist = triTriDistance2(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

template<int nDims, int nSideC, int nSideF>
__global__
void fillCutMap(dMeshBlock mb, dvec<double> cutFaces, int nCut, int nFiltF,
    dvec<int> filt_faces, int* __restrict__ cutFlag, int cutType,
    int* __restrict__ list, int ncells)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= ncells) return;

  int ic = list[tid];  // Get filtered cell ID

  // Figure out how many threads are left in this block after ic>=ncells returns
  int blockSize = min(blockDim.x, ncells - blockIdx.x * blockDim.x);

  const int nvert = nSideC*nSideC*nSideC;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;

  const bool LINEAR = (nSideC == 2 && nSideF == 2);

  //bool PRINT = threadIdx.x == 0 && blockIdx.x == 0;

  // Load up the cell nodes into an array
  double xv[nDims*nvert];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < nDims; d++)
      xv[nDims*i+d] = mb.coord[ic+mb.ncells*(d+nDims*i)]; /// NOTE: 'row-major' ZEFR layout

  double bboxC[2*nDims];
  double xcC[3];
  double xcFf[3];
  float oobb[15];
  cuda_funcs::getBoundingBox<nDims,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<nDims,nvert>(xv,xcC);

  bool PRINT = false; //(fabs(xcC[0]-.71) < .08) && (fabs(xcC[1]+.56) < .08) && (fabs(xcC[2]+.56) < .08);

  cuda_funcs::getOOBB<nDims,nvert>(xv,oobb,PRINT);

  /*float xcRc[3];
  for (int d = 0; d < 3; d++)
    xcRc[d] = 0.5f * (oobb[9+d] + oobb[12+d]);*/

  const double href = (bboxC[3]-bboxC[0]+bboxC[4]-bboxC[1]+bboxC[5]-bboxC[2]) / nDims;
  const double dtol = 2e-1*href;

  /* --- Pass 1: Coarse-Grained Distance Calculation Using Bounding Boxes --- */

  double distList[NF1];
  int faceList[NF1];
  short nHit = 0;
  __shared__ double fxv_s[nDims*4];
  float fxv_r[nDims*4] = {0.0f};

  for (int i = 0; i < NF1; i++)
  {
    distList[i] = BIG_DOUBLE;
    faceList[i] = -1;
  }

  for (int i = 0; i < nFiltF; i++)
  {
    int ff = filt_faces[i];

    __syncthreads();

    // Stick to just linear component of face [4 corners]
    for (int i = threadIdx.x; i < 4*nDims; i += blockSize)
      fxv_s[i] = cutFaces[ff*stride+i];

    __syncthreads();

    // Transform face points to element OOBB axes
    for (int i = 0; i < 12; i++)
      fxv_r[i] = 0.f;

    for (int k = 0; k < 4; k++)
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
          fxv_r[3*k+i] += oobb[3*i+j] * fxv_s[3*k+j];

    // Get OBB of face in element's axes
    float obbF[6];
    cuda_funcs::getBoundingBox<nDims,4>(fxv_r,obbF);

    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcFf);

    /*
    cuda_funcs::getCentroid<nDims,4>(fxv_s,xcF);

    double dist2 = 0.;
    for (int d = 0; d < 3; d++)
      dist2 += (xcC[d] - xcF[d]) * (xcC[d] - xcF[d]);*/

    float dist1 = cuda_funcs::boundingBoxDist<3>(&oobb[9], &obbF[0]);
    //bool check = cuda_funcs::boundingBoxCheck<3>(oobb,obbF,.01f*(float)dtol);
    bool check = (dist1 <= .01f*(float)dtol);

    double dist2 = 0;
    for (int d = 0; d < 3; d++)
      dist2 += (xcFf[d] - xcC[d]) * (xcFf[d] - xcC[d]);
    dist2 = 0.5*(sqrt(dist2) + dist1);

    if (check)
    {
      // Build up list of faces with bbox intersect from the front
      nHit++;

      int ind = min(NF1 - 1, nHit);

      if (dist2 > distList[ind]) continue;

      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > 0)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }
    else
    {
      int ind = NF1 - 1;
      distList[ind] = dist2;
      faceList[ind] = ff;
      while (distList[ind] < distList[ind-1] && ind > nHit)
      {
        swap(distList[ind], distList[ind-1]);
        swap(faceList[ind], faceList[ind-1]);
        ind--;
      }
    }
  }

  /* --- Pass 2: Coarse-Grained Intersection Check using linear portion of cell/face --- */

  int ncheck = (nHit > 0) ? min(nHit,NF1) : NF2;

  int faces[NF2];
  for (int i = 0; i < NF2; i++)
  {
    faces[i] = -1;
    distList[i] = 1e15;
  }

  double fxv[nDims*nvertf];
  double myNorm[3] = {0., 0., 0.};
  double myDist = BIG_DOUBLE;
  int myFlag = DC_UNASSIGNED;
  int nMin = 0;

  for (int F = 0; F < ncheck; F++)
  {
    if (LINEAR && myFlag == DC_CUT) continue;

    int ff = faceList[F];
    if (ff < 0)
      continue;

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    dPoint vec;
    double dist = intersectionCheck<2,2>(mb, fxv, xv, &vec[0]);
    //double dist = intersectionCheckLinear(mb, fxv, xv, &vec[0]);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (LINEAR && dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .1*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .1*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from surface; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }

    if (!LINEAR)
    {
      // Insert face into final sorted list of faces to check more fully
      if (dist < distList[NF2-1])
      {
        // Insert and sort
        distList[NF2-1] = dist;
        faces[NF2-1] = ff;
        int ind = NF2 - 1;
        while (distList[ind] < distList[ind-1] && ind > 0)
        {
          swap(distList[ind], distList[ind-1]);
          swap(faces[ind], faces[ind-1]);
          ind--;
        }
      }
    }
  }

  /* --- 6/24/17
     + Do 'intersectionCheck()' on linear component of cell/face only
     ++ Do for all ncheck faces
     + If nSideC and nSideF == 2, return this result
     + If nSideC or nSideF > 2, do 'full' intersectionCheck() using
       the nearest half of the element
     --- */

  // If entirely linear grid system, call it good & return
  if (LINEAR)
  {
    if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

    cutFlag[ic] = myFlag;
    return;
  }

  /* --- Pass 3: More-Accurate Distace Calculate Using Triangles --- */

  // Reset variables
  myDist = BIG_DOUBLE;
  myFlag = DC_UNASSIGNED;
  nMin = 0;

  for (int F = 0; F < NF2; F++)
  {
    if (myFlag == DC_CUT) continue;

    int ff = faces[F];
    if (ff < 0)
      continue;

    // Load face vertices
    for (int i = 0; i < stride; i++)
      fxv[i] = cutFaces[ff*stride+i];

    // Find distance from face to cell
    dPoint vec;
    double dist = intersectionCheck<nSideC,nSideF>(mb, fxv, xv, &vec[0]);
    vec /= vec.norm();

    dPoint norm = faceNormal(fxv);
    if (cutType == 0) norm *= -1;

    if (dist < dtol) // They intersect 1e-8*href
    {
      myFlag = DC_CUT;
      myDist = 0.;
    }
    else if (myFlag == DC_UNASSIGNED || dist < (myDist - .02*dtol))
    {
      // Unflagged cell, or have a closer face to use
      double dot = norm*vec;

      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = norm[d];

      nMin = 1;

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - myDist) <= .02*dtol)
    {
      // Approx. same dist. to two faces; avg. their normals to decide
      myDist = dist;
      for (int d = 0; d < 3; d++)
        myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.);
      nMin++;

      //myDot = norm*vec;
      double dot = 0.;
      for (int d = 0; d < 3; d++)
        dot += myNorm[d]*vec[d];

      if (dot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (myDist > 5*href)
    {
      // Clearly far from surface; cut our losses & return with what we have
      break;
    }
    else
    {
      // dist > myDist, ignore
    }
    //if (PRINT) printf("After face %d-%d: Cut Flag = %d\n",F,ff,myFlag);
  }

  if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  cutFlag[ic] = myFlag;
}

__device__
int floatToOrderedInt(float floatVal)
{
  int intVal = __float_as_int(floatVal);

  return (intVal >= 0 ) ? intVal : intVal ^ 0x8FFFFFFF;
}

__device__
unsigned int floatToUint(float fval)
{
  unsigned int ival = __float_as_uint(fval);
  unsigned int mask = -int(ival >> 31) | 0x80000000;
  return ival ^ mask;
}

__device__
float uintToFloat(unsigned int ival)
{
  unsigned int mask = ((ival >> 31) - 1) | 0x80000000;
  return __uint_as_float(ival ^ mask);
}

__device__
float orderedIntToFloat(int intVal)
{
  return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF );
}

__device__ float atomicMaxf(float* address, float val)
{
  //int *iaddr = (int*)address;
  int old = __float_as_int(*address);
  int assumed;
  while (val > __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

__device__ float atomicMinf(float* address, float val)
{
  int old = __float_as_int(*address);
  int assumed;
  while (val < __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

/*! Remove all elements which do not intersect with cut group's bbox from
 *  consideration (obviously do not intersect) */
template<int nvert>
__global__
void filterElements(dMeshBlock mb, dvec<double> cut_bbox, dvec<int> filt,
    dvec<int> cutFlag, dvec<int> nfilt, dvec<float> bboxOut)
{
  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ic == 0)
  {
    nfilt[0] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1.e10f;
      bboxOut[d+3] = -1.e10f;
    }
  }

  __shared__ float bboxF[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxF[i] = (float)cut_bbox[i];

  __syncthreads();

  if (ic >= mb.ncells) return;

  // Set all cell flags initially to DC_NORMAL (filtered cells will remain 'NORMAL')
  cutFlag[ic] = DC_NORMAL;

  float href = .005/3.*(bboxF[3]-bboxF[0]+bboxF[4]-bboxF[1]+bboxF[5]-bboxF[2]);

  // Get element nodes
  float xv[nvert*3];
  for (int i = 0; i < nvert; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = (float)mb.coord[ic+mb.ncells*(d+3*i)];

  // Get element bounding box
  float bboxC[6], xc[3];
  cuda_funcs::getBoundingBox<3,nvert>(xv, bboxC);
  cuda_funcs::getCentroid<3,nvert>(xv, xc);

  if (mb.rrot) // Transform xc to hole map's coordinate system
  {
    double x2[3] = {0.,0.,0.};
    for (int d1 = 0; d1 < 3; d1++)
      for (int d2 = 0; d2 < 3; d2++)
        x2[d1] += mb.Rmat[d1+3*d2]*(xc[d2]-mb.offset[d2]);

    for (int d = 0; d < 3; d++)
      xc[d] = x2[d];
  }

  char tag = cuda_funcs::checkHoleMap(xc, mb.hm_sam.data(), mb.hm_nx.data(), mb.hm_extents.data());
  bool checkH = (tag != 1);
  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, href);

  // If filtering element due to being completely inside hole region, tag as hole
  if (tag == 1)
    cutFlag[ic] = DC_HOLE;

  if (checkH && checkB)
  {
    int ind = atomicAggInc(&nfilt[0]);
    filt[ind] = ic;
    for (int d = 0; d < 3; d++)
    {
      atomicMinf(&bboxOut[d], bboxC[d]);
      atomicMaxf(&bboxOut[d+3], bboxC[d+3]);
    }
  }
}

/*! Remove all cutting faces which do not intersect this rank's reduced bbox
 *  from consideration (obviously do not intersect) */
template<int nvertf>
__global__
void filterFaces(dMeshBlock mb, dvec<float> ele_bbox, int nCut,
    dvec<double> cutFaces, dvec<int> filt, dvec<int> nfilt, dvec<float> bboxOut)
{
  const unsigned int ff = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ff == 0)
  {
    nfilt[1] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1e10f;
      bboxOut[d+3] = -1e10f;
    }
  }

  __shared__ float bboxE[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxE[i] = ele_bbox[i];

  __syncthreads();

  if (ff >= nCut) return;

  float href = .01f/3.f*(bboxE[3]-bboxE[0]+bboxE[4]-bboxE[1]+bboxE[5]-bboxE[2]);

  // Get face nodes
  float fxv[nvertf*3];
  for (int i = 0; i < nvertf; i++)
    for (int d = 0; d < 3; d++)
      fxv[3*i+d] = (float)cutFaces[(ff*nvertf+i)*3+d];

  // Get face bounding box
  float bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

  /// TODO: apply Rmat, offset to xc!
  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxF, bboxE, href);

  if (checkB)
  {
    int ind = atomicAggInc(&nfilt[1]);
    filt[ind] = ff;
    for (int d = 0; d < 3; d++)
    {
       atomicMinf(&bboxOut[d], bboxF[d]);
       atomicMaxf(&bboxOut[d+3], bboxF[d+3]);
    }
  }
}

void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, double *cutBbox_h, int* cutFlag, int cutType)
{
  // Setup cutMap TODO: create initialization elsewhere?
  cutFlag_d.resize(ncells);
  filt_eles.resize(ncells);
  filt_faces.resize(nCut);

  dvec<double> cutFaces;
  cutFaces.assign(cutFaces_h, nCut*nvertf*nDims);

  dvec<double> cutBbox_d;
  cutBbox_d.assign(cutBbox_h, 2*nDims);
  if (nDims != 3) printf("Bad nDims!!!! nDims = %d\n",nDims); /// DEBUGGING

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);
  }

  // Filter elements based upon cutting-surface bounding box

  hvec<int> nfilt_h;
  dvec<int> nfilt_d;
  nfilt_h.resize(2);
  nfilt_h[0] = 0;  nfilt_h[1] = 0;
  nfilt_d.assign(nfilt_h.data(), nfilt_h.size());

  ele_bbox.resize(6);
  face_bbox.resize(6);

  int threads = 128;
  int blocks = (ncells + threads - 1) / threads;

  switch(nvert)
  {
    case 8:
      filterElements<8><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox);
      break;
//    case 27:
//      filterElements<27><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
//    case 64:
//      filterElements<64><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
//    case 125:
//      filterElements<125><<<blocks, threads, 6*sizeof(double)>>>(*this, cutBbox_d, filt_list, cutFlag_d, nfilt_d);
//      break;
    default:
      printf("nvert = %d\n",nvert);
      ThrowException("nvert case not implemented for filterElements on device");
  }

  check_error();

  blocks = (nCut + threads - 1) / threads;

  switch(nvertf)
  {
    case 4:
      filterFaces<4><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    case 16:
      filterFaces<16><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d, face_bbox);
      break;
    default:
      printf("nvertf = %d\n",nvertf);
      ThrowException("nvertf case not implemented for filterFaces on device");
  }

  hipDeviceSynchronize();
  check_error();

  nfilt_h.assign(nfilt_d.data(), 2);
  int nfilt = nfilt_h[0];
  printf("nfilt = %d, %d\n",nfilt_h[0], nfilt_h[1]);

  // Perform the Direct Cut algorithm on the filtered list of grid elements

  threads = 64;
  blocks = (nfilt + threads - 1) / threads;
  int nbShare = sizeof(double)*4*nDims;

  if (nfilt > 0)
  {
    switch(nvertf)
    {
      case 4:
        switch(nvert)
        {
          case 8:
            fillCutMap<3,2,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, nfilt_h[1], filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfilt);
            break;
//          case 27:
//            fillCutMap<3,3,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,2><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
        }
        break;
//      case 9:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 27:
//            fillCutMap<3,3,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,3><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
      case 16:
        switch(nvert)
        {
          case 8:
            fillCutMap<3,2,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, nfilt_h[1], filt_faces, cutFlag_d.data(), cutType, filt_eles.data(), nfilt);
            break;
//          case 27:
//            fillCutMap<3,3,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,4><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
          default:
            printf("nvert = %d\n",nvert);
            ThrowException("nvert case not implemented for directCut on device");
        }
        break;
//      case 25:
//        switch(nvert)
//        {
//          case 8:
//            fillCutMap<3,2,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 27:
//            fillCutMap<3,3,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 64:
//            fillCutMap<3,4,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          case 125:
//            fillCutMap<3,5,5><<<blocks, threads, nbShare>>>(*this, cutFaces, nCut, cutFlag_d.data(), cutType, filt_list.data(), nfilt);
//            break;
//          default:
//            printf("nvert = %d\n",nvert);
//            ThrowException("nvert case not implemented for directCut on device");
//        }
//        break;
      default:
        printf("nvertFace = %d\n",nvertf);
        ThrowException("nvertFace case not implemented for directCut on device");
    }
  }

  hipDeviceSynchronize();
  check_error();

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  nfilt_d.free_data();
  nfilt_h.free_data();

  cutFaces.free_data();
  cutBbox_d.free_data();
}
